#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 15 "bison.y"



#include "lex.yy.c"
#include "cm.h"
#include "operators.h"




/* Line 189 of yacc.c  */
#line 84 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     NONEQUAL = 268,
     OR = 269,
     XOR = 270,
     AND = 271,
     DISTINCT = 272,
     YEAR = 273,
     MONTH = 274,
     DAY = 275,
     REGEXP = 276,
     LIKE = 277,
     IS = 278,
     IN = 279,
     NOT = 280,
     BETWEEN = 281,
     COMPARISON = 282,
     SHIFT = 283,
     MOD = 284,
     FROM = 285,
     DELETE = 286,
     LOAD = 287,
     FILTER = 288,
     BY = 289,
     JOIN = 290,
     STORE = 291,
     INTO = 292,
     GROUP = 293,
     SELECT = 294,
     AS = 295,
     ORDER = 296,
     ASC = 297,
     DESC = 298,
     COUNT = 299,
     USING = 300,
     SUM = 301,
     AVG = 302,
     MIN = 303,
     MAX = 304,
     LIMIT = 305,
     ON = 306,
     BINARY = 307,
     LEFT = 308,
     RIGHT = 309,
     OUTER = 310,
     SEMI = 311,
     ANTI = 312,
     SORT = 313,
     SEGMENTS = 314,
     PRESORTED = 315,
     PARTITION = 316,
     INSERT = 317,
     WHERE = 318,
     DISPLAY = 319,
     CASE = 320,
     WHEN = 321,
     THEN = 322,
     ELSE = 323,
     END = 324,
     SHOW = 325,
     TABLES = 326,
     TABLE = 327,
     DESCRIBE = 328,
     DROP = 329,
     CREATE = 330,
     INDEX = 331,
     INTERVAL = 332,
     APPEND = 333
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 25 "bison.y"

    long long int intval;
    double floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 207 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 219 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  23
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   928

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  96
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  14
/* YYNRULES -- Number of rules.  */
#define YYNRULES  98
/* YYNRULES -- Number of states.  */
#define YYNSTATES  294

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   333

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    25,     2,     2,     2,    36,    30,     2,
      89,    90,    34,    32,    92,    33,    91,    35,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    95,    88,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    38,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    93,    29,    94,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    21,    22,    23,    24,
      26,    27,    28,    31,    37,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63,
      64,    65,    66,    67,    68,    69,    70,    71,    72,    73,
      74,    75,    76,    77,    78,    79,    80,    81,    82,    83,
      84,    85,    86,    87
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    39,    46,
      55,    65,    73,    82,    85,    93,    99,   107,   110,   114,
     137,   146,   157,   159,   163,   165,   167,   169,   171,   173,
     175,   187,   197,   204,   207,   210,   215,   220,   225,   230,
     235,   238,   243,   248,   253,   257,   261,   265,   269,   273,
     277,   281,   285,   289,   293,   297,   301,   304,   307,   311,
     315,   321,   325,   334,   338,   343,   344,   348,   352,   358,
     360,   362,   366,   368,   372,   373,   375,   378,   383,   390,
     397,   404,   410,   416,   423,   429,   435,   443,   451,   458,
     466,   473,   481,   488,   489,   492,   493,   498,   506
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      97,     0,    -1,    98,    88,    -1,    97,    98,    88,    -1,
      99,    -1,     4,    11,    48,   102,    39,     4,   101,    -1,
       4,    11,    41,     3,    54,    89,     3,    90,    49,    89,
     103,    90,    -1,     4,    11,    42,     4,   106,    -1,     4,
      11,    50,     4,    43,   105,    -1,     4,    11,    48,   102,
      39,     4,   107,   101,    -1,    45,     4,    46,     3,    54,
      89,     3,    90,   108,    -1,    45,     4,    46,     3,   108,
      61,   109,    -1,    45,     4,    46,     3,    87,   108,    61,
     109,    -1,    82,     4,    -1,    71,    46,     4,    48,   102,
      39,     4,    -1,    40,    39,     4,    72,   100,    -1,    73,
       4,    54,    89,     3,    90,   108,    -1,    79,    80,    -1,
      83,    81,     4,    -1,    84,    85,     4,    60,     4,    89,
       4,    91,     4,    90,    39,     4,    92,     4,    72,     4,
      91,     4,    12,     4,    91,     4,    -1,    84,    85,     4,
      60,     4,    89,     4,    90,    -1,    84,    86,     4,    60,
       4,    89,     4,    92,     4,    90,    -1,     4,    -1,     4,
      91,     4,    -1,    10,    -1,     5,    -1,     6,    -1,     7,
      -1,     9,    -1,     8,    -1,     4,    93,     6,    94,    95,
       4,    89,     6,    92,     6,    90,    -1,     4,    93,     6,
      94,    95,     4,    89,     6,    90,    -1,     4,    93,     6,
      94,    95,     4,    -1,     4,    51,    -1,     4,    52,    -1,
      53,    89,   100,    90,    -1,    55,    89,   100,    90,    -1,
      56,    89,   100,    90,    -1,    57,    89,   100,    90,    -1,
      58,    89,   100,    90,    -1,    17,   100,    -1,    18,    89,
     100,    90,    -1,    19,    89,   100,    90,    -1,    20,    89,
     100,    90,    -1,   100,    32,   100,    -1,   100,    33,   100,
      -1,   100,    34,   100,    -1,   100,    35,   100,    -1,   100,
      36,   100,    -1,   100,    37,   100,    -1,   100,    16,   100,
      -1,   100,    12,   100,    -1,   100,    13,   100,    -1,   100,
      14,   100,    -1,   100,    15,   100,    -1,   100,    31,   100,
      -1,    26,   100,    -1,    25,   100,    -1,   100,    28,   100,
      -1,   100,    22,   100,    -1,   100,    28,    89,    99,    90,
      -1,    89,   100,    90,    -1,    74,    75,   100,    76,   100,
      77,   100,    78,    -1,   100,    23,     8,    -1,   100,    23,
      26,     8,    -1,    -1,    47,    43,   104,    -1,   100,    49,
       4,    -1,   102,    92,   100,    49,     4,    -1,    34,    -1,
     100,    -1,   103,    92,   100,    -1,   100,    -1,   100,    92,
     104,    -1,    -1,   104,    -1,    43,   100,    -1,    44,     4,
      60,   100,    -1,    62,    66,    44,     4,    60,   100,    -1,
      63,    66,    44,     4,    60,   100,    -1,    62,    65,    44,
       4,    60,   100,    -1,    62,    44,     4,    60,   100,    -1,
      63,    44,     4,    60,   100,    -1,    63,    65,    44,     4,
      60,   100,    -1,    64,    44,     4,    60,   100,    -1,    44,
       4,    60,   100,   107,    -1,    62,    66,    44,     4,    60,
     100,   107,    -1,    63,    66,    44,     4,    60,   100,   107,
      -1,    62,    44,     4,    60,   100,   107,    -1,    62,    65,
      44,     4,    60,   100,   107,    -1,    63,    44,     4,    60,
     100,   107,    -1,    63,    65,    44,     4,    60,   100,   107,
      -1,    64,    44,     4,    60,   100,   107,    -1,    -1,    59,
       6,    -1,    -1,    67,    68,    43,     4,    -1,    67,    68,
      43,     4,    70,    43,     6,    -1,    69,    43,     4,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   128,   128,   129,   133,   136,   138,   140,   142,   144,
     146,   148,   150,   152,   154,   156,   158,   160,   162,   164,
     166,   168,   174,   175,   176,   177,   178,   179,   180,   181,
     182,   183,   184,   185,   186,   187,   188,   189,   190,   191,
     192,   193,   194,   195,   199,   200,   201,   202,   203,   204,
     205,   206,   207,   208,   209,   210,   211,   212,   213,   214,
     216,   217,   218,   222,   223,   226,   229,   233,   234,   235,
     239,   240,   244,   245,   248,   250,   253,   257,   258,   259,
     260,   261,   262,   263,   264,   265,   266,   267,   268,   269,
     270,   271,   272,   274,   277,   279,   282,   283,   284
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL",
  "NONEQUAL", "OR", "XOR", "AND", "DISTINCT", "YEAR", "MONTH", "DAY",
  "REGEXP", "LIKE", "IS", "IN", "'!'", "NOT", "BETWEEN", "COMPARISON",
  "'|'", "'&'", "SHIFT", "'+'", "'-'", "'*'", "'/'", "'%'", "MOD", "'^'",
  "FROM", "DELETE", "LOAD", "FILTER", "BY", "JOIN", "STORE", "INTO",
  "GROUP", "SELECT", "AS", "ORDER", "ASC", "DESC", "COUNT", "USING", "SUM",
  "AVG", "MIN", "MAX", "LIMIT", "ON", "BINARY", "LEFT", "RIGHT", "OUTER",
  "SEMI", "ANTI", "SORT", "SEGMENTS", "PRESORTED", "PARTITION", "INSERT",
  "WHERE", "DISPLAY", "CASE", "WHEN", "THEN", "ELSE", "END", "SHOW",
  "TABLES", "TABLE", "DESCRIBE", "DROP", "CREATE", "INDEX", "INTERVAL",
  "APPEND", "';'", "'('", "')'", "'.'", "','", "'{'", "'}'", "':'",
  "$accept", "stmt_list", "stmt", "select_stmt", "expr", "opt_group_list",
  "expr_list", "load_list", "val_list", "opt_val_list", "opt_where",
  "join_list", "opt_limit", "sort_def", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,   276,   277,   278,   279,    33,   280,   281,   282,   124,
      38,   283,    43,    45,    42,    47,    37,   284,    94,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,   310,   311,   312,   313,   314,   315,
     316,   317,   318,   319,   320,   321,   322,   323,   324,   325,
     326,   327,   328,   329,   330,   331,   332,   333,    59,    40,
      41,    46,    44,   123,   125,    58
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    96,    97,    97,    98,    99,    99,    99,    99,    99,
      99,    99,    99,    99,    99,    99,    99,    99,    99,    99,
      99,    99,   100,   100,   100,   100,   100,   100,   100,   100,
     100,   100,   100,   100,   100,   100,   100,   100,   100,   100,
     100,   100,   100,   100,   100,   100,   100,   100,   100,   100,
     100,   100,   100,   100,   100,   100,   100,   100,   100,   100,
     100,   100,   100,   100,   100,   101,   101,   102,   102,   102,
     103,   103,   104,   104,   105,   105,   106,   107,   107,   107,
     107,   107,   107,   107,   107,   107,   107,   107,   107,   107,
     107,   107,   107,   108,   108,   109,   109,   109,   109
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     5,     6,     8,
       9,     7,     8,     2,     7,     5,     7,     2,     3,    22,
       8,    10,     1,     3,     1,     1,     1,     1,     1,     1,
      11,     9,     6,     2,     2,     4,     4,     4,     4,     4,
       2,     4,     4,     4,     3,     3,     3,     3,     3,     3,
       3,     3,     3,     3,     3,     3,     2,     2,     3,     3,
       5,     3,     8,     3,     4,     0,     3,     3,     5,     1,
       1,     3,     1,     3,     0,     1,     2,     4,     6,     6,
       6,     5,     5,     6,     5,     5,     7,     7,     6,     7,
       6,     7,     6,     0,     2,     0,     4,     7,     3
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     4,     0,     0,     0,     0,     0,    17,    13,
       0,     0,     0,     1,     0,     2,     0,     0,     0,     0,
       0,     0,     0,     0,    18,     0,     0,     3,     0,     0,
      22,    25,    26,    27,    29,    28,    24,     0,     0,     0,
       0,     0,     0,    69,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,    93,     0,     0,     0,     0,
       0,     0,     7,    33,    34,     0,     0,    40,     0,     0,
       0,    57,    56,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,    74,    15,
       0,     0,    93,     0,     0,     0,     0,     0,     0,    76,
      23,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,    61,    51,    52,    53,    54,    50,    59,    63,     0,
       0,    58,    55,    44,    45,    46,    47,    48,    49,    67,
      65,     0,    72,    75,     8,     0,    94,     0,    95,     0,
      93,     0,     0,     0,     0,    41,    42,    43,    35,    36,
      37,    38,    39,     0,    64,    22,     0,     0,     0,     0,
       0,     0,     5,    65,     0,     0,     0,    95,     0,     0,
      11,    14,    16,     0,     0,     0,     0,     0,    60,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     9,    68,
      73,    93,    12,     0,     0,    20,     0,     0,     0,    32,
       0,     0,    66,     0,     0,     0,     0,     0,     0,     0,
      10,     0,    98,     0,     0,     0,     0,     0,    77,     0,
       0,     0,     0,     0,     0,     0,    96,     0,    21,    70,
       0,     0,    62,    85,    81,     0,     0,    82,     0,     0,
      84,     0,     0,     6,     0,    31,     0,    88,    80,    78,
      90,    83,    79,    92,     0,     0,    71,     0,    89,    86,
      91,    87,    97,     0,    30,     0,     0,     0,     0,     0,
       0,     0,     0,    19
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,    10,    11,    12,   152,   182,    62,   250,   153,   154,
      72,   183,   113,   190
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -180
static const yytype_int16 yypact[] =
{
     255,     0,   -17,    21,   -15,    36,    -3,    84,    11,   -40,
     282,   -10,  -180,    49,    94,    69,   113,    64,  -180,  -180,
     116,   126,   131,  -180,    48,  -180,   134,   135,   163,   137,
      66,   139,    95,    55,  -180,    86,    88,  -180,    97,   109,
     -49,  -180,  -180,  -180,  -180,  -180,  -180,   236,    65,    67,
      68,   236,   236,  -180,    72,    73,    75,    77,    85,    78,
     236,    91,   -31,   120,   236,   -50,   163,   172,   173,   174,
      87,   236,  -180,  -180,  -180,   180,   179,   875,   236,   236,
     236,   270,   270,   236,   236,   236,   236,   236,   236,   381,
     236,   236,   236,   236,   236,   236,     9,   259,   236,   236,
     236,   236,   236,   236,   236,   182,   183,   236,   236,   789,
     101,   185,   140,   133,   -24,   106,   111,   114,   199,   789,
    -180,   110,   407,   441,   467,   493,   527,   553,   579,   613,
     692,  -180,   789,   814,   837,   859,   875,   891,  -180,   197,
      76,   309,   238,    74,    74,  -180,  -180,  -180,  -180,  -180,
     -34,   761,   355,  -180,  -180,   203,  -180,   147,   -53,   211,
     140,   218,   219,   136,   129,  -180,  -180,  -180,  -180,  -180,
    -180,  -180,  -180,   236,  -180,    -4,   142,   221,   186,   -39,
     -32,   189,  -180,   187,   231,   236,   148,   -53,   171,   204,
    -180,  -180,  -180,   -37,   156,   200,   246,   665,  -180,   191,
     236,   253,   214,   216,   276,   237,   239,   283,  -180,  -180,
    -180,   140,  -180,   245,   286,  -180,   292,   293,   210,   220,
     236,   236,  -180,   248,   307,   314,   260,   315,   317,   263,
    -180,   320,  -180,   223,   240,   236,   323,   639,   718,   236,
     271,   272,   236,   275,   287,   236,   266,   310,  -180,   789,
     -72,   -51,  -180,  -180,   718,   236,   236,   718,   236,   236,
     718,   308,   346,  -180,   236,  -180,   348,  -180,   718,   718,
    -180,   718,   718,  -180,   350,   265,   789,   262,  -180,  -180,
    -180,  -180,  -180,   354,  -180,   288,   358,   268,   359,   360,
     369,   284,   370,  -180
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -180,  -180,   366,   241,   -28,   196,   316,  -180,  -179,  -180,
    -180,   -59,  -111,   193
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_int16 yytable[] =
{
      61,   157,    73,    74,   110,   201,   210,    13,   106,   111,
     177,    13,   204,   178,   188,   159,   189,   138,   263,    77,
     264,   222,    14,    81,    82,    15,   202,   203,   179,   180,
     181,    16,    89,   205,   206,   139,   109,   112,    61,   265,
      17,   266,    75,   119,    76,    21,    22,    73,    74,   192,
     122,   123,   124,   215,   216,   125,   126,   127,   128,   129,
     130,   107,   132,   133,   134,   135,   136,   137,   107,   141,
     142,   143,   144,   145,   146,   147,   148,    18,    25,   151,
     175,    41,    42,    43,    44,    45,    46,    75,    19,    76,
      26,    27,    20,    47,    48,    49,    50,    28,    30,    29,
     230,    51,    52,    90,    91,    92,    93,    94,   101,   102,
     103,   104,    89,    95,    96,    31,     2,    32,    33,    97,
      34,     3,    98,    99,   100,   101,   102,   103,   104,    54,
      35,    55,    56,    57,    58,    36,    37,    38,    64,    39,
     105,    63,    65,    66,    67,   197,    68,     4,    69,     5,
      59,    70,    71,    88,    78,     6,    79,    80,     7,     8,
       9,    83,    84,   108,    85,    60,    86,    40,    41,    42,
      43,    44,    45,    46,    87,   115,   118,   116,   117,   253,
      47,    48,    49,    50,   120,   121,   149,   150,    51,    52,
     155,   156,   237,   238,   158,   267,   160,    53,   270,   111,
     161,   273,   163,   162,   164,   174,   186,   249,   187,   278,
     279,   254,   280,   281,   257,   191,    54,   260,    55,    56,
      57,    58,   193,   194,   196,   199,   195,   268,   269,   200,
     271,   272,   198,   207,   178,   209,   276,    59,   211,   213,
      40,    41,    42,    43,    44,    45,    46,   214,   217,   218,
     219,   221,    60,    47,    48,    49,    50,   223,   224,     1,
     225,    51,    52,    40,    41,    42,    43,    44,    45,    46,
      99,   100,   101,   102,   103,   104,    47,    48,    49,    50,
     226,   227,    23,   228,    51,    52,     1,   229,   231,    54,
     232,    55,    56,    57,    58,     2,   233,   234,    97,   235,
       3,    98,    99,   100,   101,   102,   103,   104,   239,   236,
      59,   240,    54,   247,    55,    56,    57,    58,   241,   243,
     242,   244,     2,   245,   246,    60,     4,     3,     5,   251,
     248,   255,   256,    59,     6,   258,   261,     7,     8,     9,
      98,    99,   100,   101,   102,   103,   104,   259,   140,   262,
     275,   274,   284,     4,   277,     5,   282,   283,   285,   288,
     286,     6,   287,   289,     7,     8,     9,    90,    91,    92,
      93,    94,   290,   291,   293,   292,    24,    95,    96,   208,
     212,   176,   114,    97,     0,     0,    98,    99,   100,   101,
     102,   103,   104,    90,    91,    92,    93,    94,     0,     0,
       0,     0,     0,    95,    96,     0,     0,     0,     0,    97,
       0,     0,    98,    99,   100,   101,   102,   103,   104,    90,
      91,    92,    93,    94,     0,     0,     0,     0,     0,    95,
      96,     0,     0,     0,     0,    97,     0,     0,    98,    99,
     100,   101,   102,   103,   104,     0,     0,   185,     0,     0,
       0,     0,     0,    90,    91,    92,    93,    94,     0,     0,
       0,     0,     0,    95,    96,     0,     0,     0,     0,    97,
       0,   131,    98,    99,   100,   101,   102,   103,   104,    90,
      91,    92,    93,    94,     0,     0,     0,     0,     0,    95,
      96,     0,     0,     0,     0,    97,     0,   165,    98,    99,
     100,   101,   102,   103,   104,    90,    91,    92,    93,    94,
       0,     0,     0,     0,     0,    95,    96,     0,     0,     0,
       0,    97,     0,     0,    98,    99,   100,   101,   102,   103,
     104,   166,     0,     0,     0,     0,     0,     0,     0,    90,
      91,    92,    93,    94,     0,     0,     0,     0,     0,    95,
      96,     0,     0,     0,     0,    97,     0,   167,    98,    99,
     100,   101,   102,   103,   104,    90,    91,    92,    93,    94,
       0,     0,     0,     0,     0,    95,    96,     0,     0,     0,
       0,    97,     0,   168,    98,    99,   100,   101,   102,   103,
     104,    90,    91,    92,    93,    94,     0,     0,     0,     0,
       0,    95,    96,     0,     0,     0,     0,    97,     0,     0,
      98,    99,   100,   101,   102,   103,   104,   169,     0,     0,
       0,     0,     0,     0,     0,    90,    91,    92,    93,    94,
       0,     0,     0,     0,     0,    95,    96,     0,     0,     0,
       0,    97,     0,   170,    98,    99,   100,   101,   102,   103,
     104,    90,    91,    92,    93,    94,     0,     0,     0,     0,
       0,    95,    96,     0,     0,     0,     0,    97,     0,   171,
      98,    99,   100,   101,   102,   103,   104,    90,    91,    92,
      93,    94,     0,     0,     0,     0,     0,    95,    96,     0,
       0,     0,     0,    97,     0,     0,    98,    99,   100,   101,
     102,   103,   104,   172,    90,    91,    92,    93,    94,     0,
       0,     0,     0,     0,    95,    96,     0,   252,     0,     0,
      97,     0,     0,    98,    99,   100,   101,   102,   103,   104,
      90,    91,    92,    93,    94,     0,     0,     0,     0,     0,
      95,    96,   220,     0,     0,     0,    97,     0,     0,    98,
      99,   100,   101,   102,   103,   104,     0,     0,     0,     0,
       0,     0,   177,     0,     0,     0,     0,     0,   173,     0,
       0,     0,     0,    90,    91,    92,    93,    94,     0,     0,
     179,   180,   181,    95,    96,     0,     0,     0,     0,    97,
       0,     0,    98,    99,   100,   101,   102,   103,   104,     0,
       0,    90,    91,    92,    93,    94,     0,     0,     0,     0,
     184,    95,    96,     0,     0,     0,     0,    97,     0,     0,
      98,    99,   100,   101,   102,   103,   104,    91,    92,    93,
      94,     0,     0,     0,     0,     0,    95,    96,     0,     0,
       0,     0,    97,     0,     0,    98,    99,   100,   101,   102,
     103,   104,    93,    94,     0,     0,     0,     0,     0,    95,
      96,     0,     0,     0,     0,    97,     0,     0,    98,    99,
     100,   101,   102,   103,   104,    94,     0,     0,     0,     0,
       0,    95,    96,     0,     0,     0,     0,    97,     0,     0,
      98,    99,   100,   101,   102,   103,   104,    95,    96,     0,
       0,     0,     0,    97,     0,     0,    98,    99,   100,   101,
     102,   103,   104,    -1,    -1,     0,     0,     0,     0,    97,
       0,     0,    98,    99,   100,   101,   102,   103,   104
};

static const yytype_int16 yycheck[] =
{
      28,   112,    51,    52,    54,    44,   185,    11,    39,    59,
      44,    11,    44,    47,    67,    39,    69,     8,    90,    47,
      92,   200,    39,    51,    52,     4,    65,    66,    62,    63,
      64,    46,    60,    65,    66,    26,    64,    87,    66,    90,
       4,    92,    91,    71,    93,    85,    86,    51,    52,   160,
      78,    79,    80,    90,    91,    83,    84,    85,    86,    87,
      88,    92,    90,    91,    92,    93,    94,    95,    92,    97,
      98,    99,   100,   101,   102,   103,   104,    80,    88,   107,
       4,     5,     6,     7,     8,     9,    10,    91,     4,    93,
      41,    42,    81,    17,    18,    19,    20,    48,     4,    50,
     211,    25,    26,    12,    13,    14,    15,    16,    34,    35,
      36,    37,   140,    22,    23,    46,    40,     4,    54,    28,
       4,    45,    31,    32,    33,    34,    35,    36,    37,    53,
       4,    55,    56,    57,    58,     4,    88,     3,    72,     4,
      49,     4,     3,    48,    89,   173,    60,    71,    60,    73,
      74,    54,    43,    75,    89,    79,    89,    89,    82,    83,
      84,    89,    89,    43,    89,    89,    89,     4,     5,     6,
       7,     8,     9,    10,    89,     3,    89,     4,     4,   238,
      17,    18,    19,    20,     4,     6,     4,     4,    25,    26,
      89,     6,   220,   221,    61,   254,    90,    34,   257,    59,
      89,   260,     3,    89,    94,     8,     3,   235,    61,   268,
     269,   239,   271,   272,   242,     4,    53,   245,    55,    56,
      57,    58,     4,     4,    95,     4,    90,   255,   256,    43,
     258,   259,    90,    44,    47,     4,   264,    74,    90,    68,
       4,     5,     6,     7,     8,     9,    10,    43,    92,    49,
       4,    60,    89,    17,    18,    19,    20,     4,    44,     4,
      44,    25,    26,     4,     5,     6,     7,     8,     9,    10,
      32,    33,    34,    35,    36,    37,    17,    18,    19,    20,
       4,    44,     0,    44,    25,    26,     4,     4,    43,    53,
       4,    55,    56,    57,    58,    40,     4,     4,    28,    89,
      45,    31,    32,    33,    34,    35,    36,    37,    60,    89,
      74,     4,    53,    90,    55,    56,    57,    58,     4,     4,
      60,     4,    40,    60,     4,    89,    71,    45,    73,     6,
      90,    60,    60,    74,    79,    60,    70,    82,    83,    84,
      31,    32,    33,    34,    35,    36,    37,    60,    89,    39,
       4,    43,    90,    71,     6,    73,     6,    92,     4,    91,
      72,    79,     4,     4,    82,    83,    84,    12,    13,    14,
      15,    16,    12,     4,     4,    91,    10,    22,    23,   183,
     187,   140,    66,    28,    -1,    -1,    31,    32,    33,    34,
      35,    36,    37,    12,    13,    14,    15,    16,    -1,    -1,
      -1,    -1,    -1,    22,    23,    -1,    -1,    -1,    -1,    28,
      -1,    -1,    31,    32,    33,    34,    35,    36,    37,    12,
      13,    14,    15,    16,    -1,    -1,    -1,    -1,    -1,    22,
      23,    -1,    -1,    -1,    -1,    28,    -1,    -1,    31,    32,
      33,    34,    35,    36,    37,    -1,    -1,    92,    -1,    -1,
      -1,    -1,    -1,    12,    13,    14,    15,    16,    -1,    -1,
      -1,    -1,    -1,    22,    23,    -1,    -1,    -1,    -1,    28,
      -1,    90,    31,    32,    33,    34,    35,    36,    37,    12,
      13,    14,    15,    16,    -1,    -1,    -1,    -1,    -1,    22,
      23,    -1,    -1,    -1,    -1,    28,    -1,    90,    31,    32,
      33,    34,    35,    36,    37,    12,    13,    14,    15,    16,
      -1,    -1,    -1,    -1,    -1,    22,    23,    -1,    -1,    -1,
      -1,    28,    -1,    -1,    31,    32,    33,    34,    35,    36,
      37,    90,    -1,    -1,    -1,    -1,    -1,    -1,    -1,    12,
      13,    14,    15,    16,    -1,    -1,    -1,    -1,    -1,    22,
      23,    -1,    -1,    -1,    -1,    28,    -1,    90,    31,    32,
      33,    34,    35,    36,    37,    12,    13,    14,    15,    16,
      -1,    -1,    -1,    -1,    -1,    22,    23,    -1,    -1,    -1,
      -1,    28,    -1,    90,    31,    32,    33,    34,    35,    36,
      37,    12,    13,    14,    15,    16,    -1,    -1,    -1,    -1,
      -1,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    -1,
      31,    32,    33,    34,    35,    36,    37,    90,    -1,    -1,
      -1,    -1,    -1,    -1,    -1,    12,    13,    14,    15,    16,
      -1,    -1,    -1,    -1,    -1,    22,    23,    -1,    -1,    -1,
      -1,    28,    -1,    90,    31,    32,    33,    34,    35,    36,
      37,    12,    13,    14,    15,    16,    -1,    -1,    -1,    -1,
      -1,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    90,
      31,    32,    33,    34,    35,    36,    37,    12,    13,    14,
      15,    16,    -1,    -1,    -1,    -1,    -1,    22,    23,    -1,
      -1,    -1,    -1,    28,    -1,    -1,    31,    32,    33,    34,
      35,    36,    37,    90,    12,    13,    14,    15,    16,    -1,
      -1,    -1,    -1,    -1,    22,    23,    -1,    78,    -1,    -1,
      28,    -1,    -1,    31,    32,    33,    34,    35,    36,    37,
      12,    13,    14,    15,    16,    -1,    -1,    -1,    -1,    -1,
      22,    23,    77,    -1,    -1,    -1,    28,    -1,    -1,    31,
      32,    33,    34,    35,    36,    37,    -1,    -1,    -1,    -1,
      -1,    -1,    44,    -1,    -1,    -1,    -1,    -1,    76,    -1,
      -1,    -1,    -1,    12,    13,    14,    15,    16,    -1,    -1,
      62,    63,    64,    22,    23,    -1,    -1,    -1,    -1,    28,
      -1,    -1,    31,    32,    33,    34,    35,    36,    37,    -1,
      -1,    12,    13,    14,    15,    16,    -1,    -1,    -1,    -1,
      49,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    -1,
      31,    32,    33,    34,    35,    36,    37,    13,    14,    15,
      16,    -1,    -1,    -1,    -1,    -1,    22,    23,    -1,    -1,
      -1,    -1,    28,    -1,    -1,    31,    32,    33,    34,    35,
      36,    37,    15,    16,    -1,    -1,    -1,    -1,    -1,    22,
      23,    -1,    -1,    -1,    -1,    28,    -1,    -1,    31,    32,
      33,    34,    35,    36,    37,    16,    -1,    -1,    -1,    -1,
      -1,    22,    23,    -1,    -1,    -1,    -1,    28,    -1,    -1,
      31,    32,    33,    34,    35,    36,    37,    22,    23,    -1,
      -1,    -1,    -1,    28,    -1,    -1,    31,    32,    33,    34,
      35,    36,    37,    22,    23,    -1,    -1,    -1,    -1,    28,
      -1,    -1,    31,    32,    33,    34,    35,    36,    37
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    40,    45,    71,    73,    79,    82,    83,    84,
      97,    98,    99,    11,    39,     4,    46,     4,    80,     4,
      81,    85,    86,     0,    98,    88,    41,    42,    48,    50,
       4,    46,     4,    54,     4,     4,     4,    88,     3,     4,
       4,     5,     6,     7,     8,     9,    10,    17,    18,    19,
      20,    25,    26,    34,    53,    55,    56,    57,    58,    74,
      89,   100,   102,     4,    72,     3,    48,    89,    60,    60,
      54,    43,   106,    51,    52,    91,    93,   100,    89,    89,
      89,   100,   100,    89,    89,    89,    89,    89,    75,   100,
      12,    13,    14,    15,    16,    22,    23,    28,    31,    32,
      33,    34,    35,    36,    37,    49,    39,    92,    43,   100,
      54,    59,    87,   108,   102,     3,     4,     4,    89,   100,
       4,     6,   100,   100,   100,   100,   100,   100,   100,   100,
     100,    90,   100,   100,   100,   100,   100,   100,     8,    26,
      89,   100,   100,   100,   100,   100,   100,   100,   100,     4,
       4,   100,   100,   104,   105,    89,     6,   108,    61,    39,
      90,    89,    89,     3,    94,    90,    90,    90,    90,    90,
      90,    90,    90,    76,     8,     4,    99,    44,    47,    62,
      63,    64,   101,   107,    49,    92,     3,    61,    67,    69,
     109,     4,   108,     4,     4,    90,    95,   100,    90,     4,
      43,    44,    65,    66,    44,    65,    66,    44,   101,     4,
     104,    90,   109,    68,    43,    90,    91,    92,    49,     4,
      77,    60,   104,     4,    44,    44,     4,    44,    44,     4,
     108,    43,     4,     4,     4,    89,    89,   100,   100,    60,
       4,     4,    60,     4,     4,    60,     4,    90,    90,   100,
     103,     6,    78,   107,   100,    60,    60,   100,    60,    60,
     100,    70,    39,    90,    92,    90,    92,   107,   100,   100,
     107,   100,   100,   107,    43,     4,   100,     6,   107,   107,
     107,   107,     6,    92,    90,     4,    72,     4,    91,     4,
      12,     4,    91,     4
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 133 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 137 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 139 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 141 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval));;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 143 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 145 "bison.y"
    {  emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval),0,-1); ;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 147 "bison.y"
    {  emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 149 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (7)].strval),(yyvsp[(4) - (7)].strval),0); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 151 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (8)].strval),(yyvsp[(4) - (8)].strval),1); ;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 153 "bison.y"
    {  emit_describe_table((yyvsp[(2) - (2)].strval));;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 155 "bison.y"
    {  emit_insert((yyvsp[(3) - (7)].strval), (yyvsp[(7) - (7)].strval));;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 157 "bison.y"
    {  emit_delete((yyvsp[(3) - (5)].strval));;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 159 "bison.y"
    {  emit_display((yyvsp[(2) - (7)].strval), (yyvsp[(5) - (7)].strval));;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 161 "bison.y"
    {  emit_show_tables();;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 163 "bison.y"
    {  emit_drop_table((yyvsp[(3) - (3)].strval));;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 165 "bison.y"
    {  emit_create_bitmap_index((yyvsp[(3) - (22)].strval), (yyvsp[(5) - (22)].strval), (yyvsp[(7) - (22)].strval), (yyvsp[(9) - (22)].strval), (yyvsp[(18) - (22)].strval), (yyvsp[(22) - (22)].strval));;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 167 "bison.y"
    {  emit_create_index((yyvsp[(3) - (8)].strval), (yyvsp[(5) - (8)].strval), (yyvsp[(7) - (8)].strval));;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 169 "bison.y"
    {  emit_create_interval((yyvsp[(3) - (10)].strval), (yyvsp[(5) - (10)].strval), (yyvsp[(7) - (10)].strval), (yyvsp[(9) - (10)].strval));;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 174 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 175 "bison.y"
    { emit_fieldname((yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 177 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 179 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 180 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 181 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    { emit_vardecimal((yyvsp[(1) - (11)].strval), (yyvsp[(3) - (11)].intval), (yyvsp[(6) - (11)].strval),  (yyvsp[(8) - (11)].intval), (yyvsp[(10) - (11)].intval));;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 183 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval), "", "");;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval), "", "");;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 185 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 187 "bison.y"
    { emit_count(); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 188 "bison.y"
    { emit_sum(); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 189 "bison.y"
    { emit_average(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 190 "bison.y"
    { emit_min(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 191 "bison.y"
    { emit_max(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 192 "bison.y"
    { emit_distinct(); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 193 "bison.y"
    { emit_year(); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    { emit_month(); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 195 "bison.y"
    { emit_day(); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 199 "bison.y"
    { emit_add(); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    { emit_minus(); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 201 "bison.y"
    { emit_mul(); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 202 "bison.y"
    { emit_div(); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 203 "bison.y"
    { emit("MOD"); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 204 "bison.y"
    { emit("MOD"); ;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 205 "bison.y"
    { emit_and(); ;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 206 "bison.y"
    { emit_eq(); ;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 207 "bison.y"
    { emit_neq(); ;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 208 "bison.y"
    { emit_or(); ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 209 "bison.y"
    { emit("XOR"); ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 210 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 211 "bison.y"
    { emit("NOT"); ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 212 "bison.y"
    { emit("NOT"); ;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 213 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 214 "bison.y"
    { emit_cmp(7); ;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 217 "bison.y"
    {emit("EXPR");;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 218 "bison.y"
    { emit_case(); ;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 222 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 223 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 65:

/* Line 1455 of yacc.c  */
#line 226 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 66:

/* Line 1455 of yacc.c  */
#line 229 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);;}
    break;

  case 67:

/* Line 1455 of yacc.c  */
#line 233 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 68:

/* Line 1455 of yacc.c  */
#line 234 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 69:

/* Line 1455 of yacc.c  */
#line 235 "bison.y"
    { emit_sel_name("*");;}
    break;

  case 70:

/* Line 1455 of yacc.c  */
#line 239 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 71:

/* Line 1455 of yacc.c  */
#line 240 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 72:

/* Line 1455 of yacc.c  */
#line 244 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 73:

/* Line 1455 of yacc.c  */
#line 245 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 74:

/* Line 1455 of yacc.c  */
#line 248 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 76:

/* Line 1455 of yacc.c  */
#line 253 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 77:

/* Line 1455 of yacc.c  */
#line 257 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 'I');;}
    break;

  case 78:

/* Line 1455 of yacc.c  */
#line 258 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '3');;}
    break;

  case 79:

/* Line 1455 of yacc.c  */
#line 259 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '4');;}
    break;

  case 80:

/* Line 1455 of yacc.c  */
#line 260 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '1');;}
    break;

  case 81:

/* Line 1455 of yacc.c  */
#line 261 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'S');;}
    break;

  case 82:

/* Line 1455 of yacc.c  */
#line 262 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'R');;}
    break;

  case 83:

/* Line 1455 of yacc.c  */
#line 263 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '2');;}
    break;

  case 84:

/* Line 1455 of yacc.c  */
#line 264 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'O');;}
    break;

  case 85:

/* Line 1455 of yacc.c  */
#line 265 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 'I'); ;}
    break;

  case 86:

/* Line 1455 of yacc.c  */
#line 266 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '3'); ;}
    break;

  case 87:

/* Line 1455 of yacc.c  */
#line 267 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '4'); ;}
    break;

  case 88:

/* Line 1455 of yacc.c  */
#line 268 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'L'); ;}
    break;

  case 89:

/* Line 1455 of yacc.c  */
#line 269 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '1'); ;}
    break;

  case 90:

/* Line 1455 of yacc.c  */
#line 270 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'R'); ;}
    break;

  case 91:

/* Line 1455 of yacc.c  */
#line 271 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), 'R'); ;}
    break;

  case 92:

/* Line 1455 of yacc.c  */
#line 272 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'O'); ;}
    break;

  case 93:

/* Line 1455 of yacc.c  */
#line 274 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 94:

/* Line 1455 of yacc.c  */
#line 277 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;

  case 95:

/* Line 1455 of yacc.c  */
#line 279 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 96:

/* Line 1455 of yacc.c  */
#line 282 "bison.y"
    { emit_sort((yyvsp[(4) - (4)].strval), 0); ;}
    break;

  case 97:

/* Line 1455 of yacc.c  */
#line 283 "bison.y"
    { emit_sort((yyvsp[(4) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 98:

/* Line 1455 of yacc.c  */
#line 284 "bison.y"
    { emit_presort((yyvsp[(3) - (3)].strval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2451 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 286 "bison.y"


bool scan_state;
unsigned int statement_count;

int execute_file(int ac, char **av)
{
    bool just_once  = 0;
    string script;
    process_count = 1000000000; //1GB by default
    verbose = 0;
	ssd = 0;
	delta = 0;
    total_buffer_size = 0;
	hash_seed = 100;

    for (int i = 1; i < ac; i++) {
        if(strcmp(av[i],"-l") == 0) {
            process_count = 1000000*atoff(av[i+1]);
        }
        else if(strcmp(av[i],"-v") == 0) {
            verbose = 1;
        }
        else if(strcmp(av[i],"-delta") == 0) {
            delta = 1;
        }		
        else if(strcmp(av[i],"-ssd") == 0) {
            ssd = 1;
        }		
        else if(strcmp(av[i],"-i") == 0) {
            interactive = 1;
            break;
        }
        else if(strcmp(av[i],"-s") == 0) {
            just_once = 1;
            interactive = 1;
            script = av[i+1];
        };
    };

    load_col_data(data_dict, "data.dictionary");
	tot_disk = 0;

    if (!interactive) {
        if((yyin = fopen(av[ac-1], "r")) == nullptr) {
            perror(av[ac-1]);
            exit(1);
        };

        if(yyparse()) {
            printf("SQL scan parse failed\n");
            exit(1);
        };

        scan_state = 1;
        std::clock_t start1 = std::clock();

        load_vars();
		
        statement_count = 0;
        clean_queues();
		filter_var.clear();

        yyin = fopen(av[ac-1], "r");
        PROC_FLUSH_BUF ( yyin );
        statement_count = 0;

        extern FILE *yyin;
        context = CreateCudaDevice(0, nullptr, verbose);

        if(!yyparse()) {
            if(verbose)
                cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        fclose(yyin);
        for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };

        if(verbose) {
            cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
			cout<< "disk time " << ( tot_disk / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    }
    else {
        context = CreateCudaDevice(0, nullptr, verbose);        
        if(!just_once)
            getline(cin, script);

        while (script != "exit" && script != "EXIT") {

            used_vars.clear();
            yy_scan_string(script.c_str());
            scan_state = 0;
            statement_count = 0;
            clean_queues();
            if(yyparse()) {
                printf("SQL scan parse failed \n");
                getline(cin, script);
                continue;
            };

            scan_state = 1;

            load_vars();

            statement_count = 0;
            clean_queues();
			filter_var.clear();
            yy_scan_string(script.c_str());
            std::clock_t start1 = std::clock();

            if(!yyparse()) {
                if(verbose)
                    cout << "SQL scan parse worked " <<  endl;
            };
            for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
                (*it).second->free();
            };
            varNames.clear();

            if(verbose) {
                cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
            if(!just_once)
                getline(cin, script);
            else
                script = "exit";
        };

        while(!buffer_names.empty()) {
            //delete [] buffers[buffer_names.front()];
			hipHostFree(buffers[buffer_names.front()]);
            buffer_sizes.erase(buffer_names.front());
            buffers.erase(buffer_names.front());
            buffer_names.pop();
        };
		for(auto it = index_buffers.begin(); it != index_buffers.end();it++) {
			hipHostFree(it->second);
        };

    };
    if(save_dict) {
        save_col_data(data_dict,"data.dictionary");
	};	

    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
	if(scratch.size()) {
		scratch.resize(0);
		scratch.shrink_to_fit();
	};	
	if(ranj.size()) {
		ranj.resize(0);
		ranj.shrink_to_fit();
	};	
    return 0;
}



//external c global to report errors
//char alenka_err[4048];


int alenkaExecute(char *s)
{
    YY_BUFFER_STATE bp;

    total_buffer_size = 0;
    scan_state = 0;
    load_col_data(data_dict, "data.dictionary");
    std::clock_t start;

    if(verbose)
        start = std::clock();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    int ret = yyparse();
    //printf("execute: returned [%d]\n", ret);
    if(!ret) {
        if(verbose)
            cout << "SQL scan parse worked" << endl;
    }

    scan_state = 1;
    load_vars();
    statement_count = 0;
    clean_queues();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    if(!yyparse()) {
        if(verbose)
            cout << "SQL scan parse worked " << endl;
    }
    else
        cout << "SQL scan parse failed" << endl;

    yy_delete_buffer(bp);

    // Clear Vars
    for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
        (*it).second->free();
    };
    varNames.clear();

    if(verbose)
        cout<< "statement time " <<  ( ( std::clock() - start ) / (double)CLOCKS_PER_SEC ) << endl;
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");
    return ret;
}



