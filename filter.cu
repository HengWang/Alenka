#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "filter.h"
#include "zone_map.h"
#include <iomanip>
#include <iostream> 
#include <sstream>  

struct cmp_functor_dict
{
    const unsigned long long* source;
    bool *dest;
    const unsigned int *pars;

    cmp_functor_dict(const unsigned long long int* _source, bool * _dest,  const unsigned int * _pars):
        source(_source), dest(_dest), pars(_pars) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned int idx = pars[0];
        unsigned int cmp = pars[1];
        unsigned int bits = ((unsigned int*)source)[1];
        unsigned int fit_count = ((unsigned int*)source)[0];
        unsigned int int_sz = 64;

        //find the source index
        unsigned int src_idx = i/fit_count;
        // find the exact location
        unsigned int src_loc = i%fit_count;
        //right shift the values
        unsigned int shifted = ((fit_count-src_loc)-1)*bits;
        unsigned long long int tmp = source[src_idx+2]  >> shifted;
        // set  the rest of bits to 0
        tmp	= tmp << (int_sz - bits);
        tmp	= tmp >> (int_sz - bits);
        //printf("COMP1 %llu %d \n", tmp, idx);
        if(cmp == 4) { // ==
            if(tmp == idx)
                dest[i] = 1;
            else
                dest[i] = 0;
        }
        else  { // !=
            if(tmp == idx)
                dest[i] = 0;
            else
                dest[i] = 1;
        };
    }
};


struct gpu_regex
{
    char  *source;
    char *pattern;
    bool * dest;
    const unsigned int *len;

    gpu_regex(char * _source,char * _pattern, bool * _dest,
              const unsigned int * _len):
        source(_source), pattern(_pattern), dest(_dest), len(_len) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        bool star = 0;
        int j = 0;
        char* s;
        char* p;
        char* str = source + len[0]*i;
        char* pat = pattern;

loopStart:
        for (s = str, p = pat; j < len[0] && *s; ++s, ++p, ++j) {
            switch (*p) {
            case '?':
                if (*s == '.') goto starCheck;
                break;
            case '%':
                star = 1;
                str = s, pat = p;
                do {
                    ++pat;
                }
                while (*pat == '%');
                if (!*pat) {
                    dest[i] = 1;
                    return;
                }
                goto loopStart;
            default:
                if (*s != *p)
                    goto starCheck;
                break;
            } /* endswitch */
        } /* endfor */
        while (*p == '%') ++p;
        dest[i] = !*p;
        return;

starCheck:
        if (!star) {
            dest[i] = 0;
            return;
        };
        str++;
        j++;
        goto loopStart;
    }
};



bool* filter(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, queue<unsigned int> op_nums_precision, CudaSet* a,
             unsigned int segment)
{

    stack<string> exe_type;
    stack<string> exe_value;
    stack<int_type*> exe_vectors;
	stack<unsigned int> exe_precision;
    stack<int_type> exe_nums;
    stack<bool*> bool_vectors;
    string  s1, s2, s1_val, s2_val;
    int_type n1, n2, res;
    	
    for(int i=0; !op_type.empty(); ++i, op_type.pop()) {

        string ss = op_type.front();
        //cout << endl << ss << " " <<  op_nums.size() << " " << op_nums_precision.size() << endl;

        if (ss.compare("NAME") == 0 || ss.compare("NUMBER") == 0 || ss.compare("FLOAT") == 0
                || ss.compare("STRING") == 0 || ss.compare("FIELD") == 0) {


            if (ss.compare("NUMBER") == 0) {
                exe_nums.push(op_nums.front());
                op_nums.pop();
                exe_type.push(ss);
				exe_precision.push(op_nums_precision.front());
				op_nums_precision.pop();
            }
            else if (ss.compare("NAME") == 0 || ss.compare("STRING") == 0) {
                exe_value.push(op_value.front());
                op_value.pop();
                exe_type.push(ss);
            }
            else if(ss.compare("FIELD") == 0) {
                size_t pos1 = op_value.front().find_first_of(".", 0);
                string tbl = op_value.front().substr(0,pos1);
                string field = op_value.front().substr(pos1+1, string::npos);
                op_value.pop();
                CudaSet *b = varNames.find(tbl)->second;
				auto val = b->h_columns_int[field][0];
				exe_nums.push(val);
				exe_type.push("NUMBER");
            }
        }
        else {
            if (ss.compare("MUL") == 0  || ss.compare("ADD") == 0 || ss.compare("DIV") == 0 || ss.compare("MINUS") == 0) {
                // get 2 values from the stack

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();				
				
								
				if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = get_vec(a, s1_val, exe_vectors);
										
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(a->op(t,val*24*60*60*1000,ss,1,0,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(a->op(t,val*60*60*1000,ss,1,0,0));						
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));					
								exe_vectors.push(a->op(t,val*60*1000,ss,1,0,0));						
							}											
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
									exe_vectors.push(a->op(t,val,ss,1,0,0));						
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										if (ss.compare("ADD") != 0 )
											val = -val;
										thrust::device_ptr<int_type> dev_ptr(t);
										thrust::host_vector<int_type> tt(a->mRecCount);
										thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
										
										for(int z = 0; z < a->mRecCount; z++) {
											tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
										};	
										thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
										exe_vectors.push(t);																
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											thrust::device_ptr<int_type> dev_ptr(t);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(t);																
										}	
										else {
											pos = s2_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												exe_vectors.push(a->op(t,val*1000,ss,1,0,0));						
											}
										}		
										
									};									
								};
							};

						};
					}
					
					exe_type.push("NAME");
					exe_value.push("");
					exe_precision.push(0);

				}
				else if (s2.compare("NAME") == 0 && s1.compare("STRING") == 0) {
                    s2_val = exe_value.top();
                    exe_value.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = get_vec(a, s1_val, exe_vectors);
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(a->op(t,val*24*60*60*1000,ss,0,0,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(a->op(t,val*60*60*1000,ss,0,0,0));						
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(a->op(t,val*60*1000,ss,0,0,0));						
							}		
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
									exe_vectors.push(a->op(t,val,ss,0,0,0));						
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										if (ss.compare("ADD") != 0 )
											val = -val;
										thrust::device_ptr<int_type> dev_ptr(t);
										thrust::host_vector<int_type> tt(a->mRecCount);
										thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
										
										for(int z = 0; z < a->mRecCount; z++) {
											tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
										};	
										thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
										exe_vectors.push(t);																
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											thrust::device_ptr<int_type> dev_ptr(t);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(t);																
										}		
										else {
											pos = s2_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												exe_vectors.push(a->op(t,val*1000,ss,0,0,0));						
											}											
										}		
									};									
								};
							};
							
						};
					}
					exe_type.push("NAME");
					exe_value.push("");
					exe_precision.push(0);				
				}
				
				else if (s1.compare("STRING") == 0 && s2.compare("STRING") == 0) {
				    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val, tt;
					bool reverse = 1;
					
					auto pos = s2_val.find("date()");
					if(pos != string::npos) {
						tt = curr_time;								
					}
					else {
						pos = s2_val.find("-"); //"1970-
						if(pos != string::npos) {
							struct std::tm tm;														
							tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
							tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
							tm.tm_mday = std::stoi(s2_val.substr(8,2));
							tm.tm_hour = std::stoi(s2_val.substr(11,2));
							tm.tm_min = std::stoi(s2_val.substr(14,2));
							tm.tm_sec = std::stoi(s2_val.substr(17,2));								
							#ifdef _WIN64
							tt = _mkgmtime (&tm);
							#else
							tt = timegm (&tm);
							#endif								
							tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
						}
						else {
							reverse = 0;
							pos = s1_val.find("date()");
							if(pos != string::npos) {
								tt = curr_time;								
							}
							else {
								pos = s1_val.find("-"); //"1970-
								if(pos != string::npos) {
									struct std::tm tm;														
									tm.tm_year = std::stoi(s1_val.substr(0,4))-1900;
									tm.tm_mon = std::stoi(s1_val.substr(5,2))-1;
									tm.tm_mday = std::stoi(s1_val.substr(8,2));
									tm.tm_hour = std::stoi(s1_val.substr(11,2));
									tm.tm_min = std::stoi(s1_val.substr(14,2));
									tm.tm_sec = std::stoi(s1_val.substr(17,2));								
									#ifdef _WIN64
									tt = _mkgmtime (&tm);
									#else
									tt = timegm (&tm);
									#endif								
									tt = tt*1000 + std::stoi(s1_val.substr(20,3));					
								}
							};							
						}	
					};


					
					
					pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = (add_interval(tt/1000, 0, stoi(s2_val.substr(0, pos)), 0, 0, 0, 0) - tt/1000)*1000;
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = (add_interval(tt/1000, stoi(s2_val.substr(0, pos)), 0, 0, 0, 0, 0) - tt/1000)*1000;
										}		
										else {
											pos = s1_val.find("DAY");
											if(pos != string::npos) {
												val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
											}					
											else {
												pos = s1_val.find("HOUR");
												if(pos != string::npos) {
													val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
												}											
												else {
													pos = s1_val.find("MINUTE");
													if(pos != string::npos) {
														val = stoi(s1_val.substr(0, pos)) * 60*1000;								
													}											
													else {
														pos = s1_val.find("MSECOND");
														if(pos != string::npos) {
															val = stoi(s1_val.substr(0, pos));
														}											
														else {
															pos = s1_val.find("MONTH");
															if(pos != string::npos) {
																val = stoi(s1_val.substr(0, pos));
																val = (add_interval(tt/1000, 0, val, 0, 0, 0, 0) - tt/1000)*1000;
															}						
															else {
																pos = s1_val.find("YEAR");
																if(pos != string::npos) {
																	val = stoi(s1_val.substr(0, pos));
																	val = (add_interval(tt/1000, val, 0, 0, 0, 0, 0)- tt/1000)*1000;
																}	
																else {
																	pos = s2_val.find("SECOND");
																	if(pos != string::npos) {
																		val = stoi(s2_val.substr(0, pos))*1000;
																	}	
																	else {
																		pos = s1_val.find("SECOND");
																		if(pos != string::npos) {
																			val = stoi(s1_val.substr(0, pos))*1000;
																		}	
																	}		
																}			
															};									
														};
													};							
												};
											}
										}	
									};									
								};
							};
						}	
					};				

				
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+tt;
                    else {
						if(!reverse)
							res = val-tt;
						else
							res = tt- val;
					};	

                    exe_type.push("NUMBER");
                    exe_nums.push(res);		
					exe_precision.push(0);					
				}				
				
				else if (s1.compare("STRING") == 0 && s2.compare("NUMBER") == 0) {
				    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
                    n1 = exe_nums.top();
                    exe_nums.pop();

					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s1_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;								
								}																		
							}
						};
					};
					
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+n1;
                    else
                        res = val-n1;

                    exe_type.push("NUMBER");
                    exe_nums.push(res);						
					exe_precision.push(0);					
				}
				
				else if (s2.compare("STRING") == 0 && s1.compare("NUMBER") == 0) {
				    s1_val = exe_value.top();
                    exe_value.pop();
					int_type val;
                    n1 = exe_nums.top();
                    exe_nums.pop();

					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos)) * 24*60*60*1000;
					}					
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos)) * 60*60*1000;
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos)) * 60*1000;								
							}											
							else {
								pos = s1_val.find("date()");
								if(pos != string::npos) {
									val = curr_time;								
								}																		
							}
						};
					};
					
					int_type res;
                    if (ss.compare("ADD") == 0 )
                        res = val+n1;
                    else
                        res = n1-val;

                    exe_type.push("NUMBER");
                    exe_nums.push(res);						
					exe_precision.push(0);					
				}
				
				else if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = get_vec(a, s1_val, exe_vectors);
					
					auto pos = s2_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s2_val.substr(0, pos));
						exe_vectors.push(a->op(t,val*24*60*60*1000,ss,1,0,0));
					}					
					else {
						pos = s2_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s2_val.substr(0, pos));
							exe_vectors.push(a->op(t,val*60*60*1000,ss,1,0,0));						
						}											
						else {
							pos = s2_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s2_val.substr(0, pos));
								exe_vectors.push(a->op(t,val*60*1000,ss,1,0,0));						
							}				
							else {
								pos = s2_val.find("MSECOND");
								if(pos != string::npos) {
									val = stoi(s2_val.substr(0, pos));
									exe_vectors.push(a->op(t,val,ss,1,0,0));						
								}											
								else {
									pos = s2_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s2_val.substr(0, pos));
										if (ss.compare("ADD") != 0 )
											val = -val;
										thrust::device_ptr<int_type> dev_ptr(t);
										thrust::host_vector<int_type> tt(a->mRecCount);
										thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
										
										for(int z = 0; z < a->mRecCount; z++) {
											tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
										};	
										thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
										exe_vectors.push(t);																
									}						
									else {
										pos = s2_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s2_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											thrust::device_ptr<int_type> dev_ptr(t);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(t);																
										}							
										else {
											pos = s2_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s2_val.substr(0, pos));
												exe_vectors.push(a->op(t,val*1000,ss,1,0,0));						
											}											
										}
									};									
								};
							};
	
						};
					}
					exe_type.push("NAME");
					exe_value.push("");
					exe_precision.push(0);
				}
				else if (s2.compare("NAME") == 0 && s1.compare("STRING") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					int_type val;
					int_type* t = get_vec(a, s2_val, exe_vectors);
					
					auto pos = s1_val.find("DAY");
					if(pos != string::npos) {
						val = stoi(s1_val.substr(0, pos));
						exe_vectors.push(a->op(t,val*24*60*60*1000,ss,0,0,0));
					}
					else {
						pos = s1_val.find("HOUR");
						if(pos != string::npos) {
							val = stoi(s1_val.substr(0, pos));
							exe_vectors.push(a->op(t,val*60*60*1000,ss,0,0,0));						
						}											
						else {
							pos = s1_val.find("MINUTE");
							if(pos != string::npos) {
								val = stoi(s1_val.substr(0, pos));
								exe_vectors.push(a->op(t,val*60*1000,ss,0,0,0));						
							}	
							else {
								pos = s1_val.find("SECOND");
								if(pos != string::npos) {
									val = stoi(s1_val.substr(0, pos));
									exe_vectors.push(a->op(t,val*1000,ss,0,0,0));						
								}											
								else {
									pos = s1_val.find("MONTH");
									if(pos != string::npos) {
										val = stoi(s1_val.substr(0, pos));
										if (ss.compare("ADD") != 0 )
											val = -val;
										thrust::device_ptr<int_type> dev_ptr(t);
										thrust::host_vector<int_type> tt(a->mRecCount);
										thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
										
										for(int z = 0; z < a->mRecCount; z++) {
											tt[z] = add_interval(tt[z], 0, val, 0, 0, 0, 0);																					
										};	
										thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
										exe_vectors.push(t);																
									}						
									else {
										pos = s1_val.find("YEAR");
										if(pos != string::npos) {
											val = stoi(s1_val.substr(0, pos));
											int_type* temp = (int_type*)malloc(2*int_size);
											if (ss.compare("ADD") != 0 )
												val = -val;											
											thrust::device_ptr<int_type> dev_ptr(t);
											thrust::host_vector<int_type> tt(a->mRecCount);
											thrust::copy(dev_ptr, dev_ptr+a->mRecCount, tt.begin());									
											
											for(int z = 0; z < a->mRecCount; z++) {
												tt[z] = add_interval(tt[z], val, 0, 0, 0, 0, 0);																					
											};	
											thrust::copy(tt.begin(), tt.end(), dev_ptr);																				
											exe_vectors.push(t);																
										}							
										else {
											pos = s1_val.find("SECOND");
											if(pos != string::npos) {
												val = stoi(s1_val.substr(0, pos));
												exe_vectors.push(a->op(t,val,ss,0,0,0));						
											}											
										}
									};									
								};
							};							
						};						
					}					
					exe_type.push("NAME");
					exe_value.push("");
					exe_precision.push(0);
				}
                else if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
					
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();					
					auto pres = precision_func(p1, p2, ss);	
					exe_precision.push(pres);
					if(p1) 
						n1 = n1*(unsigned int)pow(10,p1);
					if(p2) 
						n2 = n2*(unsigned int)pow(10,p2);

                    if (ss.compare("ADD") == 0 )
                        res = n1+n2;
                    else if (ss.compare("MUL") == 0 )
                        res = n1*n2;
                    else if (ss.compare("DIV") == 0 )
                        res = n1/n2;
                    else
                        res = n1-n2;

                    thrust::device_ptr<int_type> p = thrust::device_malloc<int_type>(a->mRecCount);
                    thrust::sequence(p, p+(a->mRecCount),res,(int_type)0);

                    exe_type.push("NAME");
					exe_value.push("");
                    exe_vectors.push(thrust::raw_pointer_cast(p));
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    n1 = exe_nums.top();
                    exe_nums.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = get_decimals(a, s1_val, exe_precision);					

					int_type* t = get_vec(a, s1_val, exe_vectors);
					auto pres = precision_func(p1, p2, ss);	
					exe_precision.push(pres);
					exe_type.push("NAME");
					exe_value.push("");
					exe_vectors.push(a->op(t,n1,ss,1, p1, p2));

                }
                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = get_decimals(a, s2_val, exe_precision);					

					int_type* t = get_vec(a, s2_val, exe_vectors);
					auto pres = precision_func(p2, p1, ss);	
					exe_precision.push(pres);
					exe_type.push("NAME");
					exe_value.push("");
					exe_vectors.push(a->op(t,n1,ss,0, p2, p1));

                }
                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if (a->type[s1_val] == 0) {
                        int_type* t1 = get_vec(a, s1_val, exe_vectors);
						int_type* t = get_vec(a, s2_val, exe_vectors);
						auto p1 = get_decimals(a, s1_val, exe_precision);					
						auto p2 = get_decimals(a, s2_val, exe_precision);												
						auto pres = precision_func(p1, p2, ss);	
						exe_precision.push(pres);
						exe_type.push("NAME");
						exe_value.push("");
						exe_vectors.push(a->op(t,t1,ss,0,p2,p1));
                    }
                }				
            }

            else if (ss.compare("CMP") == 0) {


                int_type cmp_type = op_nums.front();
                op_nums.pop();

                s1 = exe_type.top();
                exe_type.pop();
                s2 = exe_type.top();
                exe_type.pop();

                if (s1.compare("NUMBER") == 0 && s2.compare("NUMBER") == 0) {
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    n2 = exe_nums.top();
                    exe_nums.pop();
                    exe_type.push("NAME");
					exe_value.push("");
					auto p1 = exe_precision.top();
					exe_precision.pop();
					auto p2 = exe_precision.top();
					exe_precision.pop();					
					auto pres = precision_func(p1, p2, ss);	
					exe_precision.push(pres);				
					if(p1) 
						n1 = n1*(unsigned int)pow(10,p1);
					if(p2) 
						n2 = n2*(unsigned int)pow(10,p2);
					
                    bool_vectors.push(a->compare(n1,n2,cmp_type));
                }
                else if ((s1.compare("STRING") == 0 && s2.compare("NAME") == 0) ||
                         (s1.compare("NAME") == 0 && s2.compare("STRING") == 0))
                {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
					
					if(s1_val == "date()" || s2_val == "date()") {
						if(s1.compare("STRING") == 0) {
							s1_val = exe_value.top();
							exe_value.pop();
							s2_val = exe_value.top();
							exe_value.pop();
							int_type val;
							int_type* t = get_vec(a, s1_val, exe_vectors);
							
							auto pos = s2_val.find("date()");
							if(pos != string::npos) {
								val = curr_time;
								bool_vectors.push(a->compare(t,val,cmp_type,0,0));
							}		
							else {
								pos = s2_val.find("-"); //"1970-
								if(pos != string::npos) {
									struct std::tm tm;														
									tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
									tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
									tm.tm_mday = std::stoi(s2_val.substr(8,2));
									tm.tm_hour = std::stoi(s2_val.substr(11,2));
									tm.tm_min = std::stoi(s2_val.substr(14,2));
									tm.tm_sec = std::stoi(s2_val.substr(17,2));								
									#ifdef _WIN64
									auto tt = _mkgmtime (&tm);
									#else
									auto tt = timegm (&tm);
									#endif								
									tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
									bool_vectors.push(a->compare(t,tt, cmp_type,0,0));
								}
							};
							
							exe_type.push("NAME");
							exe_value.push("");
							exe_precision.push(0);												
						}
						else {
							s2_val = exe_value.top();
							exe_value.pop();
							s1_val = exe_value.top();
							exe_value.pop();
							int_type val;
							int_type* t = get_vec(a, s1_val, exe_vectors);
							
							auto pos = s2_val.find("date()");
							if(pos != string::npos) {
								val = curr_time;
								bool_vectors.push(a->compare(t,val, cmp_type,0,0));
							}					
							else {
								pos = s2_val.find("-"); //"1970-
								if(pos != string::npos) {
									struct std::tm tm;														
									tm.tm_year = std::stoi(s2_val.substr(0,4))-1900;
									tm.tm_mon = std::stoi(s2_val.substr(5,2))-1;
									tm.tm_mday = std::stoi(s2_val.substr(8,2));
									tm.tm_hour = std::stoi(s2_val.substr(11,2));
									tm.tm_min = std::stoi(s2_val.substr(14,2));
									tm.tm_sec = std::stoi(s2_val.substr(17,2));								
									#ifdef _WIN64
									auto tt = _mkgmtime (&tm);
									#else
									auto tt = timegm (&tm);
									#endif								
									tt = tt*1000 + std::stoi(s2_val.substr(20,3));					
									bool_vectors.push(a->compare(t,tt, cmp_type,0,0));
								}
							};
							exe_type.push("NAME");
							exe_value.push("");
							exe_precision.push(0);					
						}						
					}
					else {	

						if (s1.compare("NAME") == 0 && s2.compare("STRING") == 0) {
							s1.swap(s2);
							s1_val.swap(s2_val);
						};
						
						if (a->type[s2_val] == 0 && a->ts_cols[s2_val] ) {
							struct std::tm tm;						
							auto year = s1_val.substr(0,4);
							auto month = s1_val.substr(5,2);
							auto day = s1_val.substr(8,2);
							auto hour = s1_val.substr(11,2);
							auto min = s1_val.substr(14,2);
							auto sec = s1_val.substr(17,2);
							auto usec = s1_val.substr(20,3);
							//cout << "VL " << year << " " << month << " " << day << " " << hour << " " << min << " " << sec << " " << usec << "   " << endl;
							
							tm.tm_year = std::stoi(year)-1900;
							tm.tm_mon = std::stoi(month)-1;
							tm.tm_mday = std::stoi(day);
							tm.tm_hour = std::stoi(hour);
							tm.tm_min = std::stoi(min);
							tm.tm_sec = std::stoi(sec);
							
							#ifdef _WIN64
							time_t time = _mkgmtime (&tm);
							#else
							time_t time = timegm (&tm);
							#endif
							
							
							time = time*1000 + std::stoi(usec);					
							int_type* t = get_vec(a, s2_val, exe_vectors);
							exe_precision.push(0);						
							exe_type.push("NAME");
							exe_value.push("");
							bool_vectors.push(a->compare(t,(int_type)time,cmp_type, 0, 0));						

						}
						else {	
						
							void* d_res, *d_v;
							if(cmp_type != 7)
								hipMalloc((void **) &d_res, a->mRecCount);
							else
								hipMalloc((void **) &d_res, a->hostRecCount);
							thrust::device_ptr<bool> dd_res((bool*)d_res);

							hipMalloc((void **) &d_v, 8);
							thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
							thrust::counting_iterator<unsigned int> begin(0);

							if(s2_val.find(".") != string::npos) { //bitmap index
								auto pos1 = s2_val.find_first_of(".");
								auto pos2 = s2_val.find_last_of(".");
								auto set = s2_val.substr(pos1+1, (pos2-pos1)-1);
								auto col = s2_val.substr(pos2+1);
								auto len = data_dict[set][col].col_length;								
								
								while(s1_val.length() < len)
									s1_val = s1_val + '\0';

								auto s1_hash = MurmurHash64A(&s1_val[0], len, hash_seed)/2;								
								if(a->idx_dictionary_int[s2_val].find(s1_hash) != a->idx_dictionary_int[s2_val].end()) {
									dd_v[0] = a->idx_dictionary_int[s2_val][s1_hash];
									dd_v[1] = (unsigned int)cmp_type;
									cmp_functor_dict ff(idx_vals[s2_val], (bool*)d_res, (unsigned int*)d_v);
									thrust::for_each(begin, begin + a->mRecCount, ff);
								}
								else {
									hipMemset(d_res,0,a->mRecCount);
								}
							}
							else {

								auto s = a->string_map[s2_val];
								auto pos = s.find_first_of(".");
								auto len = data_dict[s.substr(0, pos)][s.substr(pos+1)].col_length;

								dd_v[0] = len;
								dd_v[1] = (unsigned int)s1_val.length() + 1;

								if(cmp_type != 7) {
									thrust::device_vector<unsigned long long int> vv(1);
									while(s1_val.length() < len) {
										s1_val = s1_val + '\0';
									};

									vv[0] = MurmurHash64A(&s1_val[0], s1_val.length(), hash_seed)/2;

									string f1 = a->load_file_name + "." + s2_val + "." + to_string(segment) + ".hash";
									FILE* f = fopen(f1.c_str(), "rb" );
									unsigned long long int* buff = new unsigned long long int[a->mRecCount];
									unsigned int cnt;
									fread(&cnt, 4, 1, f);
									fread(buff, a->mRecCount*8, 1, f);
									fclose(f);
									thrust::device_vector<unsigned long long int> vals(a->mRecCount);
									thrust::copy(buff, buff+a->mRecCount, vals.begin());
									if(cmp_type == 4) //==
										thrust::transform(vals.begin(), vals.end(), thrust::make_constant_iterator(vv[0]), dd_res, thrust::equal_to<unsigned long long int>());
									else if(cmp_type == 3) //!=
										thrust::transform(vals.begin(), vals.end(), thrust::make_constant_iterator(vv[0]), dd_res, thrust::not_equal_to<unsigned long long int>());
									delete [] buff;

								}
								else {
									if(a->map_like.find(s2_val) == a->map_like.end()) {
								
										void* d_str;
										hipMalloc((void **) &d_str, len);
										hipMemset(d_str,0,len);
										hipMemcpy( d_str, (void *) s1_val.c_str(), s1_val.length(), hipMemcpyHostToDevice);
										
										string f1 = a->load_file_name + "." + s2_val;
										FILE* f = fopen(f1.c_str(), "rb" );
										fseek(f, 0, SEEK_END);
										long fileSize = ftell(f);
										fseek(f, 0, SEEK_SET);																
																		
										unsigned int pieces = 1;
										if(fileSize > getFreeMem()/2)
											pieces = fileSize /(getFreeMem()/2) + 1;
										auto piece_sz = fileSize/pieces;
										ldiv_t ldivresult = ldiv(fileSize/pieces, len);		
										if(ldivresult.rem != 0)
											piece_sz = fileSize/pieces + (len - ldivresult.rem);										
										thrust::device_vector<char> dev(piece_sz);	
										char* buff = new char[piece_sz];
										a->map_res[s2_val] = thrust::device_vector<unsigned int>();
										for(auto i = 0; i < pieces; i++) {	
											
											if(i == pieces-1)
												piece_sz = fileSize - piece_sz*i;											
											fread(buff, piece_sz, 1, f);	
											hipMemcpy( thrust::raw_pointer_cast(dev.data()), (void*)buff, piece_sz, hipMemcpyHostToDevice);

											gpu_regex ff(thrust::raw_pointer_cast(dev.data()), (char*)d_str, (bool*)d_res, (unsigned int*)d_v);
											thrust::for_each(begin, begin + piece_sz/len, ff);
											
											auto cnt = thrust::count(dd_res, dd_res + piece_sz/len, 1);
											auto offset = a->map_res[s2_val].size();
											a->map_res[s2_val].resize(a->map_res[s2_val].size() + cnt);
											thrust::copy_if(thrust::make_counting_iterator((unsigned int)(i*(piece_sz/len))), thrust::make_counting_iterator((unsigned int)((i+1)*(piece_sz/len))),
															dd_res, a->map_res[s2_val].begin() + offset, thrust::identity<bool>());
										};				
										
										fclose(f);
										delete [] buff;												
										hipFree(d_str);
										thrust::sort(a->map_res[s2_val].begin(), a->map_res[s2_val].end());
										a->map_like[s2_val] = 1;


									};
									// now lets calc the current segments's matches
									hipMemset(d_res, 0, a->hostRecCount);
									binary_search(a->map_res[s2_val].begin(),a->map_res[s2_val].end(), a->d_columns_int[s2_val].begin(), a->d_columns_int[s2_val].end(), dd_res);
								};
							};

							hipFree(d_v);
							exe_type.push("NAME");
							bool_vectors.push((bool*)d_res);
						}	
					}
                }

                else if (s1.compare("NUMBER") == 0 && s2.compare("NAME") == 0) {					
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s1_val = exe_value.top();
                    exe_value.pop();

                    if(s1_val.find(".") != string::npos) { //bitmap index
                        void* d_v, *d_res;
                        hipMalloc((void **) &d_v, 8);
                        thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                        hipMalloc((void **) &d_res, a->mRecCount);
                        if(a->idx_dictionary_int[s1_val].find(n1) != a->idx_dictionary_int[s1_val].end()) {
                            dd_v[0] = a->idx_dictionary_int[s1_val][n1];
                            dd_v[1] = (unsigned int)cmp_type;
                            thrust::counting_iterator<unsigned int> begin(0);
                            cmp_functor_dict ff(idx_vals[s1_val], (bool*)d_res, (unsigned int*)d_v);
                            thrust::for_each(begin, begin + a->mRecCount, ff);							
                        }
                        else {
                            hipMemset(d_res,0,a->mRecCount);
                        };
                        exe_type.push("NAME");
                        bool_vectors.push((bool*)d_res);
                        hipFree(d_v);
                    }
                    else {
						int_type* t = get_vec(a, s1_val, exe_vectors);
						thrust::device_ptr<int_type> bp((int_type*)t);
						auto p2 = exe_precision.top();
						exe_precision.pop();
						auto p1 = get_decimals(a, s1_val, exe_precision);			
						auto pres = std::max(p1, p2);	
						exe_precision.push(pres);
						
						exe_type.push("NAME");
						bool_vectors.push(a->compare(t,n1,cmp_type, pres-p1, pres-p2));
                    };
                }
                else if (s1.compare("NAME") == 0 && s2.compare("NUMBER") == 0) {
                    cmp_type = reverse_op(cmp_type);
                    n1 = exe_nums.top();
                    exe_nums.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();

                    if(s2_val.find(".") != string::npos) { //bitmap index
                        void* d_v, *d_res;
                        hipMalloc((void **) &d_v, 8);
                        thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
                        hipMalloc((void **) &d_res, a->mRecCount);

                        if(a->idx_dictionary_int[s2_val].find(n1) != a->idx_dictionary_int[s2_val].end()) {

                            dd_v[0] = a->idx_dictionary_int[s2_val][n1];
                            dd_v[1] = (unsigned int)cmp_type;

                            thrust::counting_iterator<unsigned int> begin(0);
                            cmp_functor_dict ff(idx_vals[s2_val], (bool*)d_res, (unsigned int*)d_v);
                            thrust::for_each(begin, begin + a->mRecCount, ff);
                        }
                        else {
                            hipMemset(d_res,0,a->mRecCount);
                        };
                        exe_type.push("NAME");
                        bool_vectors.push((bool*)d_res);
                        hipFree(d_v);
                    }
                    else {
						int_type* t = get_vec(a, s2_val, exe_vectors);
						auto p2 = exe_precision.top();
						exe_precision.pop();
						auto p1 = get_decimals(a, s2_val, exe_precision);					
						auto pres = std::max(p1, p2);	
						exe_precision.push(pres);							
						exe_type.push("NAME");
						bool_vectors.push(a->compare(t,n1,cmp_type, p1, p2));
                    };
                }

                else if (s1.compare("NAME") == 0 && s2.compare("NAME") == 0) {
                    s1_val = exe_value.top();
                    exe_value.pop();
                    s2_val = exe_value.top();
                    exe_value.pop();
                    exe_type.push("NAME");

                    int_type* t = get_vec(a, s1_val, exe_vectors);
					int_type* t1 = get_vec(a, s2_val, exe_vectors);
					auto p1 = get_decimals(a, s1_val, exe_precision);					
					auto p2 = get_decimals(a, s2_val, exe_precision);					
					auto pres = max(p1, p2);	
					exe_precision.push(pres);
					bool_vectors.push(a->compare(t1,t,cmp_type, p2, p1));
				}
			}	

            else if (ss.compare("AND") == 0) {
                bool* s3 = bool_vectors.top();
                bool_vectors.pop();
                bool* s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("NAME");
                bool_vectors.push(a->logical_and(s2,s3));
            }
            else if (ss.compare("OR") == 0) {
                bool* s3 = bool_vectors.top();
                bool_vectors.pop();
                bool* s2 = bool_vectors.top();
                bool_vectors.pop();
                exe_type.push("NAME");
                bool_vectors.push(a->logical_or(s2,s3));
            }
            else {
                cout << "found nothing " << endl;
            }
        };
    };

    return bool_vectors.top();
}