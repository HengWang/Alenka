#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <iostream>
#include <ctime>
#include <string>

#include "alenka.h"

using namespace std;
using namespace alenka;

int main(int ac, char **av) {
    std::clock_t start;
    int x;
    // test QPS via alenkaExecute	-- this section is the only C++ dependency
    if (ac == 2 && string(av[1]) == "--QPS-test") {
        init(NULL);
        start = std::clock();
        for (x=0; x< 1000; x++)  {
            execute("A1 := SELECT  count(n_name) AS col1 FROM nation;\n DISPLAY A1 USING ('|');");
        }
        LOG(alenka::logINFO) << "Ave QPS is : " <<  (1000/ ((std::clock() - start) / (double)CLOCKS_PER_SEC));
        close();
    } else {  // ordinary alenka file mode
        if (ac < 2) {
            cerr << "Usage : alenka [--QPS-test] | [ [-l load size(MB)] [-v] script.sql ]" << endl;
            exit(EXIT_FAILURE);
        } else {
            return execute_file(ac, av);
        }
    }
}


