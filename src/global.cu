/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <map>
#include <queue>
#include <string>
#include <vector>

#include "global.h"

namespace alenka {

//config
string data_dict_local_name = "data.dictionary";
string data_dict_redis_simple_host = "localhost";
unsigned int data_dict_redis_simple_port = 6379;
bool data_dict_redis_simple_pool = true;
string data_dict_redis_ha_sentinel_host = "redis1,redis2,redis3";
string data_dict_redis_ha_master_name = "redis-cluster";
unsigned int data_dict_redis_ha_sentinel_port = 26379;
bool data_dict_redis_ha_pool = true;
string file_system_local_base_path= ".";
string file_system_hdfs_host = "default";
unsigned int file_system_hdfs_port = 0;
string file_system_hdfs_base_path = "/tmp/";

//common
size_t int_size = sizeof(int_type);
size_t float_size = sizeof(float_type);
unsigned int total_segments = 0, old_segments;
unsigned int hash_seed;
time_t curr_time;
void* d_v = nullptr;
void* s_v = nullptr;
queue<string> op_type;
bool op_case;
queue<string> op_sort;
queue<string> op_presort;
queue<string> op_value;
string grp_val;
queue<int_type> op_nums;
queue<float_type> op_nums_f;
queue<unsigned int> op_nums_precision; //decimals' precision
queue<string> col_aliases;
size_t total_count = 0, oldCount, total_max, totalRecs, alloced_sz = 0;
size_t process_count;
bool fact_file_loaded = 1;
unsigned int partition_count;
map<string, string> setMap; //map to keep track of column names and set names
clock_t tot;
clock_t tot_disk;
unsigned long long int currtime;
bool verbose;
bool save_dict = 0;
bool interactive;
bool ssd;
bool delta;
bool star;
map<string, char*> index_buffers;
map<string, char*> buffers;
map<string, size_t> buffer_sizes;
queue<string> buffer_names;
size_t total_buffer_size;
thrust::device_vector<unsigned char> scratch;
char* readbuff;
thrust::device_vector<unsigned int> rcol_matches;
thrust::device_vector<int_type> rcol_dev;
thrust::device_vector<int> ranj;
size_t allocated_sz;
standard_context_t context;
map<unsigned int, map<unsigned long long int, size_t> > char_hash; // mapping between column's string hashes and string positions
bool scan_state;
unsigned int statement_count;
map<string, map<string, bool> > used_vars;
map<string, unsigned int> cpy_bits;
map<string, long long int> cpy_init_val;
bool phase_copy;
map<string, bool> min_max_eq;
map<string, string> filter_var;
void* alloced_tmp;
bool alloced_switch = 0;
vector<void*> alloced_mem;
IDataDict *data_dict;
map<string, unsigned long long int*> idx_vals; // pointer to compressed values in gpu memory
map<string, unsigned int> cnt_counts;
string curr_file;
IFileSystem *file_system;
stringstream display_results;

struct res{
	int code;
	string results;
};

//operators
queue<string> namevars;
queue<string> typevars;
queue<int> sizevars;
queue<int> cols;
queue<unsigned int> j_col_count;
unsigned int sel_count = 0;
unsigned int join_cnt = 0;;
unsigned int distinct_cnt = 0;
unsigned int join_col_cnt = 0;
unsigned int join_tab_cnt = 0;
unsigned int tab_cnt = 0;
queue<string> op_join;
queue<char> join_type;
queue<char> join_eq_type;
map<string, unsigned int> stat;
map<unsigned int, unsigned int> join_and_cnt;

int get_utc_offset() {
	time_t zero = 24*60*60L;
	struct tm * timeptr;
	int gmtime_hours;

	/* get the local time for Jan 2, 1900 00:00 UTC */
	timeptr = localtime(&zero);
	gmtime_hours = timeptr->tm_hour;

	/* if the local time is the "day before" the UTC, subtract 24 hours
	from the hours to get the UTC offset */
	if (timeptr->tm_mday < 2)
	gmtime_hours -= 24;

	return gmtime_hours;
}

/*
  the utc analogue of mktime,
  (much like timegm on some systems)
*/
time_t tm_to_time_t_utc(struct tm * timeptr) {
  /* gets the epoch time relative to the local time zone,
  and then adds the appropriate number of seconds to make it UTC */
  return mktime(timeptr) + get_utc_offset() * 3600;
}

size_t getFreeMem() {
    size_t available, total;
    hipMemGetInfo(&available, &total);
    return available;
}

#ifdef _WIN64
size_t getTotalSystemMemory() {
    MEMORYSTATUSEX status;
    status.dwLength = sizeof(status);
    GlobalMemoryStatusEx(&status);
    return status.ullTotalPhys;
}
#else
size_t getTotalSystemMemory() {
    long pages = sysconf(_SC_PHYS_PAGES);
    long page_size = sysconf(_SC_PAGE_SIZE);
    return pages * page_size;
}
#endif

void process_error(int severity, string err) {
    switch (severity) {
    case 1:
        err = "(Warning) " + err;
        break;
    case 2:
        err = "(Fatal) " + err;
        break;
    default:
        err = "(Aborting) " + err;
        break;
    }
    error_cb(severity, err.c_str());            // send the error to the c based callback
}

time_t add_interval(time_t t, int year, int month, int day, int hour, int minute, int second) {
	if (year) {
		struct tm tt = *gmtime(&t);
		tt.tm_year = tt.tm_year + year;
		return tm_to_time_t_utc(&tt);
	} else if (month) {
		struct tm tt = *gmtime(&t);
		if (tt.tm_mon + month > 11) {
			tt.tm_year++;
			tt.tm_mon = ((tt.tm_mon + month) - 11)-1;
		} else {
			tt.tm_mon = tt.tm_mon + month;
		}
		return tm_to_time_t_utc(&tt);
	} else if (day) {
		return t + day*24*60*60;
	} else if (hour) {
		return t + hour*60*60;
	} else if (minute) {
		return t + minute*60;
	} else {
		return t + second;
	}
}

void alloc_pool(unsigned int maxRecs) {
	void* temp;
	CUDA_SAFE_CALL(hipMalloc((void **) &temp, 8*maxRecs));
	alloced_mem.push_back(temp);
}


} // namespace alenka

