#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <iostream>
#include <ctime>
#include <string>

#include "alenka.h"

using namespace std;

int main(int ac, char **av) {
    std::clock_t start;

    //overwrite config - see global.cu for defaults
    alenka::data_dict_local_name = "alenka.dictonary";

    // test QPS via alenka::execute
    if (ac == 2 && string(av[1]) == "--QPS-test") {
    	alenka::init(NULL);
        start = std::clock();
        for (int x=0; x< 1000; x++)  {
        	alenka::execute("A1 := SELECT  count(n_name) AS col1 FROM nation;\n DISPLAY A1 USING ('|');");
        }
        cout << "Ave QPS is : " <<  (1000/ ((std::clock() - start) / (double)CLOCKS_PER_SEC)) << endl;
        alenka::close();
    } else {  // ordinary alenka::execute_file file mode
        if (ac < 2) {
            cerr << "Usage : alenka [--QPS-test] | [ [-l load size(MB)] [-v] script.sql ]" << endl;
            exit(EXIT_FAILURE);
        } else {
            return alenka::execute_file(ac, av);
        }
    }
}


