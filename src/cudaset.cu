#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <queue>
#include <string>
#include <map>
#include <stack>
#include <set>
#include <vector>
#include <algorithm>

#include "cudaset.h"

namespace alenka {

extern void copyFinalize(CudaSet* a, queue<string> fields, bool ts);
extern void update_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, string SortType, char* tmp, unsigned int len);
extern void apply_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, char* res, unsigned int len);
extern void allocColumns(CudaSet* a, queue<string> fields);
extern void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz = 0, bool flt = 1);
extern map<string, CudaSet*> varNames; //  STL map to manage CudaSet variables

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs)
    : mColumnCount(0), mRecCount(0) {
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs);
    source = 1;
    text_source = 1;
    fil_f = nullptr;
    fil_s = nullptr;
}

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name, unsigned int max)
    : mColumnCount(0),  mRecCount(0) {
    maxRecs = max;
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
    source = 1;
    text_source = 0;
    fil_f = nullptr;
    fil_s = nullptr;
}

CudaSet::CudaSet(const size_t RecordCount, const unsigned int ColumnCount) {
    initialize(RecordCount, ColumnCount);
    keep = false;
    source = 0;
    text_source = 0;
    fil_f = nullptr;
    fil_s = nullptr;
}

CudaSet::CudaSet(queue<string> op_sel, const queue<string> op_sel_as) {
    initialize(op_sel, op_sel_as);
    keep = false;
    source = 0;
    text_source = 0;
    fil_f = nullptr;
    fil_s = nullptr;
}

CudaSet::CudaSet(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as) {
    initialize(a, b, op_sel, op_sel_as);
    keep = false;
    source = 0;
    text_source = 0;
    fil_f = nullptr;
    fil_s = nullptr;
}

CudaSet::~CudaSet() {
    free();
}

void CudaSet::allocColumnOnDevice(string colname, size_t RecordCount) {
    if (type[colname] != 1) {
        d_columns_int[colname].resize(RecordCount);
    } else {
        d_columns_float[colname].resize(RecordCount);
    }
}

void CudaSet::resize_join(size_t addRecs) {
    mRecCount = mRecCount + addRecs;
    for (unsigned int i=0; i < columnNames.size(); i++) {
        if (type[columnNames[i]] != 1) {
            h_columns_int[columnNames[i]].resize(mRecCount);
        } else {
            h_columns_float[columnNames[i]].resize(mRecCount);
        }
    }
}

void CudaSet::resize(size_t addRecs) {
    mRecCount = mRecCount + addRecs;
    for (unsigned int i=0; i < columnNames.size(); i++) {
        if (type[columnNames[i]] != 1) {
            h_columns_int[columnNames[i]].resize(mRecCount);
        } else {
            h_columns_float[columnNames[i]].resize(mRecCount);
        }
    }
}

void CudaSet::deAllocColumnOnDevice(string colname) {
    if (type[colname] != 1 && !d_columns_int.empty() && d_columns_int.find(colname) != d_columns_int.end()) {
        if (d_columns_int[colname].size() > 0) {
            d_columns_int[colname].resize(0);
            d_columns_int[colname].shrink_to_fit();
        }
    } else if (type[colname] == 1 && !d_columns_float.empty()) {
        if (d_columns_float[colname].size() > 0) {
            d_columns_float[colname].resize(0);
            d_columns_float[colname].shrink_to_fit();
        }
    }
}

void CudaSet::allocOnDevice(size_t RecordCount) {
    for (unsigned int i=0; i < columnNames.size(); i++)
        allocColumnOnDevice(columnNames[i], RecordCount);
}

void CudaSet::deAllocOnDevice() {
    for (unsigned int i=0; i < columnNames.size(); i++) {
        deAllocColumnOnDevice(columnNames[i]);
	}

	if (prm_d.size()) {
		prm_d.resize(0);
		prm_d.shrink_to_fit();
	}

    for (auto it=d_columns_int.begin(); it != d_columns_int.end(); ++it) {
        if (it->second.size() > 0) {
            it->second.resize(0);
            it->second.shrink_to_fit();
        }
    }

    for (auto it=d_columns_float.begin(); it != d_columns_float.end(); ++it) {
        if (it->second.size() > 0) {
            it->second.resize(0);
            it->second.shrink_to_fit();
        }
    }

    if (filtered) { // dealloc the source
        if (varNames.find(source_name) != varNames.end()) {
            varNames[source_name]->deAllocOnDevice();
        }
    }
}

void CudaSet::resizeDeviceColumn(size_t RecCount, string colname) {
    if (type[colname] != 1) {
        d_columns_int[colname].resize(RecCount);
    } else {
        d_columns_float[colname].resize(RecCount);
    }
}

void CudaSet::resizeDevice(size_t RecCount) {
    for (unsigned int i=0; i < columnNames.size(); i++) {
        resizeDeviceColumn(RecCount, columnNames[i]);
    }
}

bool CudaSet::onDevice(string colname) {
    if (type[colname] != 1) {
        if (!d_columns_int.empty() && d_columns_int[colname].size())
            return 1;
    } else {
        if (!d_columns_float.empty() && d_columns_float[colname].size())
            return 1;
    }
    return 0;
}

CudaSet* CudaSet::copyDeviceStruct() {
    CudaSet* a = new CudaSet(mRecCount, mColumnCount);
    a->not_compressed = not_compressed;
    a->segCount = segCount;
    a->maxRecs = maxRecs;
    a->columnNames = columnNames;
	a->ts_cols = ts_cols;
    a->cols = cols;
    a->type = type;
    a->char_size = char_size;
    a->decimal = decimal;
	a->decimal_zeroes = decimal_zeroes;

    for (unsigned int i=0; i < columnNames.size(); i++) {
        if (a->type[columnNames[i]] == 0) {
            a->d_columns_int[columnNames[i]] = thrust::device_vector<int_type>();
            a->h_columns_int[columnNames[i]] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >();
        } else if (a->type[columnNames[i]] == 1) {
            a->d_columns_float[columnNames[i]] = thrust::device_vector<float_type>();
            a->h_columns_float[columnNames[i]] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >();
        } else {
            a->h_columns_char[columnNames[i]] = nullptr;
            a->d_columns_char[columnNames[i]] = nullptr;
        }
    }
    a->load_file_name = load_file_name;
    a->mRecCount = 0;
    return a;
}

int_type CudaSet::readSsdSegmentsFromFile(unsigned int segNum, string colname, size_t offset, thrust::host_vector<unsigned int>& prm_vh, CudaSet* dest) {
    string f1 = load_file_name + "." + colname + "." + to_string(segNum);
    iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
    if (!f) {
    	LOG(logERROR) << "Error opening " << f1 << " file ";
        exit(0);
    }

    unsigned int cnt, bits;
    int_type lower_val;

    unsigned short int val_s_r[4096/2];
    char val_c_r[4096];
    unsigned int val_i_r[4096/4];
    unsigned long long int val_l_r[4096/8];
    unsigned int idx;
    bool idx_set = 0;

    file_system->read(&cnt, 4, f);
    file_system->read(&lower_val, 8, f);
    file_system->seek(f, cnt - (8+4) + 32, SEEK_CUR);
    file_system->read(&bits, 4, f);
    LOG(logDEBUG) << "lower_val bits " << lower_val << " " << bits << endl;

    if (type[colname] == 0) {
    	LOG(logDEBUG) << "lower_val bits " << lower_val << " " << bits;
        for (unsigned int i = 0; i < prm_vh.size(); i++) {
            if (!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
            	file_system->seek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
                idx = prm_vh[i];
                idx_set = 1;

                if (bits == 8) {
                	file_system->read(&val_c_r[0], 4096, f);
                    dest->h_columns_int[colname][i + offset] = val_c_r[0];
                } else if (bits == 16) {
                	file_system->read(&val_s_r, 4096, f);
                    dest->h_columns_int[colname][i + offset] = val_s_r[0];
                }
                if (bits == 32) {
                	file_system->read(&val_i_r, 4096, f);
                    dest->h_columns_int[colname][i + offset] = val_i_r[0];
                }
                if (bits == 84) {
                	file_system->read(&val_l_r, 4096, f);
                    dest->h_columns_int[colname][i + offset] = val_l_r[0];
                }
            } else {
                if (bits == 8) {
                    dest->h_columns_int[colname][i + offset] = val_c_r[prm_vh[i]-idx];
                } else if (bits == 16) {
                    dest->h_columns_int[colname][i + offset] = val_s_r[prm_vh[i]-idx];
                }
                if (bits == 32) {
                    dest->h_columns_int[colname][i + offset] = val_i_r[prm_vh[i]-idx];
                }
                if (bits == 84) {
                    dest->h_columns_int[colname][i + offset] = val_l_r[prm_vh[i]-idx];
                }
            }
        }
    } else if (type[colname] == 1) {
        for (unsigned int i = 0; i < prm_vh.size(); i++) {
            if (!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
            	file_system->seek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
                idx = prm_vh[i];
                idx_set = 1;
                file_system->read(val_c_r, 4096, f);
                memcpy(&dest->h_columns_float[colname][i + offset], &val_c_r[0], bits/8);
            } else {
                memcpy(&dest->h_columns_float[colname][i + offset], &val_c_r[(prm_vh[i]-idx)*(bits/8)], bits/8);
            }
        }
    } else {
        //no strings in fact tables
    }
    file_system->close(f);
    return lower_val;
}

int_type CudaSet::readSsdSegmentsFromFileR(unsigned int segNum, string colname, thrust::host_vector<unsigned int>& prm_vh, thrust::host_vector<unsigned int>& dest) {
    string f1 = load_file_name + "." + colname + "." + to_string(segNum);
    iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
    if (!f) {
    	LOG(logERROR) << "Error opening " << f1 << " file " << endl;
        exit(0);
    }

    unsigned int cnt, bits;
    int_type lower_val;
    file_system->read(&cnt, 4, f);
    file_system->read(&lower_val, 8, f);
    file_system->seek(f, cnt - (8+4) + 32, SEEK_CUR);
    file_system->read(&bits, 4, f);

    unsigned short int val_s_r[4096/2];
    char val_c_r[4096];
    unsigned int val_i_r[4096/4];
    unsigned long long int val_l_r[4096/8];
    unsigned int idx;
    bool idx_set = 0;

    for (unsigned int i = 0; i < prm_vh.size(); i++) {
        if (!idx_set ||  prm_vh[i] >= idx + 4096/(bits/8))  {
        	file_system->seek(f, 24 + prm_vh[i]*(bits/8), SEEK_SET);
            idx = prm_vh[i];
            idx_set = 1;

            if (bits == 8) {
            	file_system->read(val_c_r, 4096, f);
                dest[i] = val_c_r[0];
            } else if (bits == 16) {
            	file_system->read(val_s_r, 4096, f);
                dest[i] = val_s_r[0];
            }
            if (bits == 32) {
            	file_system->read(val_i_r, 4096, f);
                dest[i] = val_i_r[0];
            }
            if (bits == 84) {
            	file_system->read(val_l_r, 4096, f);
                dest[i] = val_l_r[0];
            }
        } else {
            if (bits == 8) {
                dest[i] = val_c_r[prm_vh[i]-idx];
            } else if (bits == 16) {
                dest[i] = val_s_r[prm_vh[i]-idx];
            }
            if (bits == 32) {
                dest[i] = val_i_r[prm_vh[i]-idx];
            }
            if (bits == 84) {
                dest[i] = val_l_r[prm_vh[i]-idx];
            }
        }
    }
    file_system->close(f);
    return lower_val;
}

extern std::clock_t tot_disk;

void CudaSet::readSegmentsFromFile(unsigned int segNum, string colname) {
    string f1 = load_file_name + "." + colname + "." + to_string(segNum);
    if (type[colname] == 2)
        f1 = f1 + ".idx";

    std::clock_t start1 = std::clock();

    if (interactive) { //check if data are in buffers
        if (buffers.find(f1) == buffers.end()) { // add data to buffers
            iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
            if (!f) {
            	process_error(3, "Error opening " + string(f1) +" file ");
            }
            file_system->seek(f, 0, SEEK_END);
            long fileSize = file_system->tell(f);
            while (total_buffer_size + fileSize > getTotalSystemMemory() && !buffer_names.empty()) { //free some buffers
                //delete [] buffers[buffer_names.front()];
				hipHostFree(buffers[buffer_names.front()]);
                total_buffer_size = total_buffer_size - buffer_sizes[buffer_names.front()];
                buffer_sizes.erase(buffer_names.front());
                buffers.erase(buffer_names.front());
                buffer_names.pop();
            }
            file_system->seek(f, 0, SEEK_SET);

			char* buff;
			hipHostAlloc((void**) &buff, fileSize, hipHostMallocDefault);
			file_system->read(buff, fileSize, f);
			file_system->close(f);
            buffers[f1] = buff;
            buffer_sizes[f1] = fileSize;
            buffer_names.push(f1);
            total_buffer_size = total_buffer_size + fileSize;
            buffer_names.push(f1);
            LOG(logDEBUG) << "added buffer " << f1 << " " << fileSize << endl;
        }
        // get data from buffers
        if (type[colname] != 1) {
            unsigned int cnt = ((unsigned int*)buffers[f1])[0];
            if (cnt > h_columns_int[colname].size()/8 + 10)
                h_columns_int[colname].resize(cnt/8 + 10);
        } else {
            unsigned int cnt = ((unsigned int*)buffers[f1])[0];
            if (cnt > h_columns_float[colname].size()/8 + 10)
                h_columns_float[colname].resize(cnt/8 + 10);
        }
    } else {
    	iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
        if (!f) {
        	LOG(logERROR) << "Error opening " << f1 << " file " << endl;
            exit(0);
        }

        if (type[colname] != 1) {
            if (1 > h_columns_int[colname].size())
                h_columns_int[colname].resize(1);
            file_system->read(h_columns_int[colname].data(), 4, f);
            unsigned int cnt = ((unsigned int*)(h_columns_int[colname].data()))[0];
            if (cnt/8+10 > h_columns_int[colname].size()) {
                h_columns_int[colname].resize(cnt + 10);
			}
            size_t rr = file_system->read((unsigned int*)(h_columns_int[colname].data()) + 1, cnt+52, f);
            if (rr != cnt+52) {
                char buf[1024];
                sprintf(buf, "Couldn't read %d bytes from %s ,read only", cnt+52, f1.c_str());
                process_error(3, string(buf));
            }
        } else  {
            if (1 > h_columns_float[colname].size())
                h_columns_float[colname].resize(1);
            file_system->read(h_columns_float[colname].data(), 4, f);
            unsigned int cnt = ((unsigned int*)(h_columns_float[colname].data()))[0];
            if (cnt/8+10 > h_columns_float[colname].size())
                h_columns_float[colname].resize(cnt + 10);
            size_t rr = file_system->read((unsigned int*)(h_columns_float[colname].data()) + 1, cnt+52, f);
            if (rr != cnt+52) {
                char buf[1024];
                sprintf(buf, "Couldn't read %d bytes from %s ,read only", cnt+52, f1.c_str());
                process_error(3, string(buf));
            }
        }
        file_system->close(f);
    }
    tot_disk =  tot_disk + (std::clock() - start1);
}

void CudaSet::CopyColumnToGpu(string colname,  unsigned int segment, size_t offset) {
    if (not_compressed) {
        // calculate how many records we need to copy
        if (segment < segCount-1) {
            mRecCount = maxRecs;
        } else {
            mRecCount = hostRecCount - maxRecs*(segCount-1);
        }

        if (type[colname] != 1) {
            if (!alloced_switch) {
                thrust::copy(h_columns_int[colname].begin() + maxRecs*segment, h_columns_int[colname].begin() + maxRecs*segment + mRecCount, d_columns_int[colname].begin() + offset);
			} else {
                thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
                thrust::copy(h_columns_int[colname].begin() + maxRecs*segment, h_columns_int[colname].begin() + maxRecs*segment + mRecCount, d_col);
            }
        } else {
            if (!alloced_switch) {
                thrust::copy(h_columns_float[colname].begin() + maxRecs*segment, h_columns_float[colname].begin() + maxRecs*segment + mRecCount, d_columns_float[colname].begin() + offset);
            } else {
                thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
                thrust::copy(h_columns_float[colname].begin() + maxRecs*segment, h_columns_float[colname].begin() + maxRecs*segment + mRecCount, d_col);
            }
        }
    } else {
        readSegmentsFromFile(segment, colname);
        if (!d_v)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
        if (!s_v)
            CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

        string f1;
        if (type[colname] == 2) {
            f1 = load_file_name + "." + colname + "." + to_string(segment) + ".idx";
        } else {
            f1 = load_file_name + "." + colname + "." + to_string(segment);
        }

        if (type[colname] != 1) {
            if (!alloced_switch) {
                if (buffers.find(f1) == buffers.end()) {
                    mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + offset), h_columns_int[colname].data(), d_v, s_v, colname);
                } else {
                    mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + offset), buffers[f1], d_v, s_v, colname);
                }
            } else {
                if (buffers.find(f1) == buffers.end()) {
                    mRecCount = pfor_decompress(alloced_tmp, h_columns_int[colname].data(), d_v, s_v, colname);
                } else {
                    mRecCount = pfor_decompress(alloced_tmp, buffers[f1], d_v, s_v, colname);
                }
            }
        } else {
            if (decimal[colname]) {
                if (!alloced_switch) {
                    if (buffers.find(f1) == buffers.end()) {
                        mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_float[colname].data() + offset) , h_columns_float[colname].data(), d_v, s_v, colname);
                    } else {
                        mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_float[colname].data() + offset) , buffers[f1], d_v, s_v, colname);
                    }
					if (!phase_copy) {
						thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[colname].data() + offset));
						thrust::transform(d_col_int, d_col_int+mRecCount, d_columns_float[colname].begin(), long_to_float());
					}
                } else {
                    if (buffers.find(f1) == buffers.end()) {
                        mRecCount = pfor_decompress(alloced_tmp, h_columns_float[colname].data(), d_v, s_v, colname);
                    } else {
                        mRecCount = pfor_decompress(alloced_tmp, buffers[f1], d_v, s_v, colname);
                    }
					if (!phase_copy) {
						thrust::device_ptr<long long int> d_col_int((long long int*)alloced_tmp);
						thrust::device_ptr<float_type> d_col_float((float_type*)alloced_tmp);
						thrust::transform(d_col_int, d_col_int+mRecCount, d_col_float, long_to_float());
					}
					//for(int i = 0; i < mRecCount;i++)
					//cout << "DECOMP " << (float_type)(d_col_int[i]) << " " << d_col_float[i] << endl;
                }
            }
            //else // uncompressed float
            // will have to fix it later so uncompressed data will be written by segments too
        }
    }
}

void CudaSet::CopyColumnToGpu(string colname) { // copy all segments
    if (not_compressed) {
        if (type[colname] != 1)
            thrust::copy(h_columns_int[colname].begin(), h_columns_int[colname].begin() + mRecCount, d_columns_int[colname].begin());
        else
            thrust::copy(h_columns_float[colname].begin(), h_columns_float[colname].begin() + mRecCount, d_columns_float[colname].begin());
    } else {
        if (!d_v)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
        if (!s_v)
            CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

        size_t cnt = 0;
        string f1;

        for (unsigned int i = 0; i < segCount; i++) {
            readSegmentsFromFile(i, colname);

            if (type[colname] == 2) {
                f1 = load_file_name + "." + colname + "." + to_string(i) + ".idx";
            } else {
                f1 = load_file_name + "." + colname + "." + to_string(i);
            }

            if (type[colname] == 0) {
                if (buffers.find(f1) == buffers.end()) {
                    mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + cnt), h_columns_int[colname].data(), d_v, s_v, colname);
                } else {
                    mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[colname].data() + cnt), buffers[f1], d_v, s_v, colname);
                }

            } else if (type[colname] == 1) {
                if (decimal[colname]) {
                    if (buffers.find(f1) == buffers.end()) {
                        mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt) , h_columns_float[colname].data(), d_v, s_v, colname);
                    } else {
                        mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt) , buffers[f1], d_v, s_v, colname);
                    }
					if (!phase_copy) {
						thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[colname].data() + cnt));
						thrust::transform(d_col_int, d_col_int+mRecCount, d_columns_float[colname].begin() + cnt, long_to_float());
					}
                }
                // else  uncompressed float
                // will have to fix it later so uncompressed data will be written by segments too
            }
            cnt = cnt + mRecCount;

            //totalRecs = totals + mRecCount;
        }

        mRecCount = cnt;
    }
}

void CudaSet::CopyColumnToHost(string colname, size_t offset, size_t RecCount) {
    if (type[colname] != 1) {
        thrust::copy(d_columns_int[colname].begin(), d_columns_int[colname].begin() + RecCount, h_columns_int[colname].begin() + offset);
	} else {
        thrust::copy(d_columns_float[colname].begin(), d_columns_float[colname].begin() + RecCount, h_columns_float[colname].begin() + offset);
	}
}

void CudaSet::CopyColumnToHost(string colname) {
    CopyColumnToHost(colname, 0, mRecCount);
}

void CudaSet::CopyToHost(size_t offset, size_t count) {
    for (unsigned int i = 0; i < columnNames.size(); i++) {
        CopyColumnToHost(columnNames[i], offset, count);
    }
}

float_type* CudaSet::get_float_type_by_name(string name) {
    return thrust::raw_pointer_cast(d_columns_float[name].data());
}

int_type* CudaSet::get_int_by_name(string name) {
    return thrust::raw_pointer_cast(d_columns_int[name].data());
}

float_type* CudaSet::get_host_float_by_name(string name) {
    return thrust::raw_pointer_cast(h_columns_float[name].data());
}

int_type* CudaSet::get_host_int_by_name(string name) {
    return thrust::raw_pointer_cast(h_columns_int[name].data());
}

void CudaSet::GroupBy(stack<string> columnRef) {
    if (grp.size() < mRecCount)
        grp.resize(mRecCount);
	thrust::fill(grp.begin(), grp.begin()+mRecCount, 0);
	if (scratch.size() < mRecCount)
		scratch.resize(mRecCount*sizeof(bool));
	thrust::device_ptr<bool> d_group((bool*)thrust::raw_pointer_cast(scratch.data()));

    d_group[mRecCount-1] = 1;

    for (int i = 0; i < columnRef.size(); columnRef.pop()) {
		if (ts_cols[columnRef.top()]) {
			queue<string> fields;
			fields.push(columnRef.top());
			copyFinalize(this, fields, 1);
			time_t start_t;
			std::vector<time_t> rcol;

			thrust::device_vector<int_type> unq(mRecCount);
			thrust::copy(d_columns_int[columnRef.top()].begin(), d_columns_int[columnRef.top()].begin() + mRecCount, unq.begin());
			auto result_end = thrust::unique(unq.begin(), unq.end());

			if(unq[0] != 0 || mRecCount == 1) {
				start_t = unq[0]/1000;
			} else {
				start_t = unq[1]/1000;
			}
			time_t end_t = unq[(result_end-unq.begin())-1]/1000;

			LOG(logDEBUG) << "start end " << start_t << " " << end_t;
			//int year_start, year_end, month_start, month_end, day_start, day_end, hour_start, hour_end, minute_start, minute_end, second_start, second_end;
			//struct tm my_tm, my_tm1;
			auto my_tm = *gmtime(&start_t);
			auto my_tm1 = *gmtime(&end_t);

			LOG(logDEBUG) << my_tm.tm_year << " " << my_tm1.tm_year << " " << my_tm.tm_min << " " << my_tm1.tm_min << " " << my_tm.tm_hour << " " << my_tm1.tm_hour;
			rcol.push_back(0);//1970/01/01

			auto pos = grp_val.find("YEAR");
			int grp_num;
			if (pos != string::npos) {
				grp_num = stoi(grp_val.substr(0, pos));
				my_tm.tm_mon = 0;
				my_tm.tm_mday = 1;
				my_tm.tm_hour = 0;
				my_tm.tm_min = 0;
				my_tm.tm_sec = 0;
				start_t = tm_to_time_t_utc(&my_tm);
				rcol.push_back(start_t*1000);
				while (start_t <= end_t) {
					start_t = add_interval(start_t, grp_num, 0, 0, 0, 0, 0);
					rcol.push_back(start_t*1000);
				}
			} else {
				pos = grp_val.find("MONTH");
				int grp_num;
				if (pos != string::npos) {
					grp_num = stoi(grp_val.substr(0, pos));
					my_tm.tm_mday = 1;
					my_tm.tm_hour = 0;
					my_tm.tm_min = 0;
					my_tm.tm_sec = 0;
					start_t = tm_to_time_t_utc(&my_tm);
					LOG(logDEBUG) << "interval " << start_t;
					rcol.push_back(start_t*1000);
					while (start_t <= end_t) {
						start_t = add_interval(start_t, 0, grp_num, 0, 0, 0, 0);
						LOG(logDEBUG) << "interval " << start_t;
						rcol.push_back(start_t*1000);
					}
				} else {
					pos = grp_val.find("DAY");
					int grp_num;
					if (pos != string::npos) {
						grp_num = stoi(grp_val.substr(0, pos));
						my_tm.tm_hour = 0;
						my_tm.tm_min = 0;
						my_tm.tm_sec = 0;
						start_t = tm_to_time_t_utc(&my_tm);
						rcol.push_back(start_t*1000);
						while (start_t <= end_t) {
							start_t = add_interval(start_t, 0, 0, grp_num, 0, 0, 0);
							rcol.push_back(start_t*1000);
						}
					} else {
						pos = grp_val.find("HOUR");
						int grp_num;
						if (pos != string::npos) {
							grp_num = stoi(grp_val.substr(0, pos));
							my_tm.tm_min = 0;
							my_tm.tm_sec = 0;
							start_t = tm_to_time_t_utc(&my_tm);
							rcol.push_back(start_t*1000);
							while (start_t <= end_t) {
								start_t = add_interval(start_t, 0, 0, 0, grp_num, 0, 0);
								rcol.push_back(start_t*1000);
							}
						} else {
							pos = grp_val.find("MINUTE");
							int grp_num;
							if (pos != string::npos) {
								grp_num = stoi(grp_val.substr(0, pos));
								my_tm.tm_sec = 0;
								start_t = tm_to_time_t_utc(&my_tm);
								rcol.push_back(start_t*1000);
								while (start_t <= end_t) {
									start_t = add_interval(start_t, 0, 0, 0, 0, grp_num, 0);
									rcol.push_back(start_t*1000);
								}
							} else {
								pos = grp_val.find("SECOND");
								int grp_num;
								if (pos != string::npos) {
									grp_num = stoi(grp_val.substr(0, pos));
									start_t = tm_to_time_t_utc(&my_tm);
									rcol.push_back(start_t*1000);
									while (start_t <= end_t) {
										start_t = add_interval(start_t, 0, 0, 0, 0, 0, grp_num);
										rcol.push_back(start_t*1000);
									}
								}
							}
						}
					}
				}
			}

			//thrust::device_vector<unsigned int> output(mRecCount);
			rcol_matches.resize(mRecCount);
			rcol_dev.resize(rcol.size());
			thrust::copy(rcol.data(), rcol.data() + rcol.size(), rcol_dev.begin());
			thrust::lower_bound(rcol_dev.begin(), rcol_dev.end(), d_columns_int[columnRef.top()].begin(), d_columns_int[columnRef.top()].begin() + mRecCount, rcol_matches.begin());

			thrust::transform(rcol_matches.begin(), rcol_matches.begin() + mRecCount - 1, rcol_matches.begin()+1, d_group, thrust::not_equal_to<unsigned int>());
			thrust::transform(rcol_matches.begin(), rcol_matches.begin() + mRecCount, rcol_matches.begin(), decrease());
			d_group[mRecCount-1] = 1;
		} else {
			unsigned int bits;
			if (cpy_bits.empty())
				bits = 0;
			else
				bits = cpy_bits[columnRef.top()];

			if (bits == 8) {
				if (type[columnRef.top()] != 1) {  // int_type
					thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
					thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned char>());
				} else {
					thrust::device_ptr<unsigned char> src((unsigned char*)thrust::raw_pointer_cast(d_columns_float[columnRef.top()].data()));
					thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned char>());
				}
			} else if (bits == 16) {
				if (type[columnRef.top()] != 1) {  // int_type
					thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
					thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned short int>());
				} else {
					thrust::device_ptr<unsigned short int> src((unsigned short int*)thrust::raw_pointer_cast(d_columns_float[columnRef.top()].data()));
					thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned short int>());
				}
			} else if (bits == 32) {
				if (type[columnRef.top()] != 1) {  // int_type
					thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(d_columns_int[columnRef.top()].data()));
					thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned int>());
				} else {
					thrust::device_ptr<unsigned int> src((unsigned int*)thrust::raw_pointer_cast(d_columns_float[columnRef.top()].data()));
					thrust::transform(src, src + mRecCount - 1,	src+1, d_group, thrust::not_equal_to<unsigned int>());
				}
			} else {
				if (type[columnRef.top()] != 1) {  // int_type
					thrust::transform(d_columns_int[columnRef.top()].begin(), d_columns_int[columnRef.top()].begin() + mRecCount - 1,
						d_columns_int[columnRef.top()].begin()+1, d_group, thrust::not_equal_to<int_type>());
				} else {
					thrust::transform(d_columns_float[columnRef.top()].begin(), d_columns_float[columnRef.top()].begin() + mRecCount - 1,
								  d_columns_float[columnRef.top()].begin()+1, d_group, f_not_equal_to());
				}
			}
		}
        thrust::transform(d_group, d_group+mRecCount, grp.begin(), grp.begin(), thrust::logical_or<bool>());
    }
    grp_count = thrust::count(grp.begin(), grp.begin()+mRecCount, 1);
}

void CudaSet::addDeviceColumn(int_type* col, string colname, size_t recCount) {
    if (std::find(columnNames.begin(), columnNames.end(), colname) == columnNames.end()) {
        columnNames.push_back(colname);
        type[colname] = 0;
        d_columns_int[colname] = thrust::device_vector<int_type>(recCount);
        h_columns_int[colname] = thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >(recCount);
    } else {  // already exists, my need to resize it
        if (d_columns_int[colname].size() < recCount) {
            d_columns_int[colname].resize(recCount);
        }
		if (h_columns_int[colname].size() < recCount) {
            h_columns_int[colname].resize(recCount);
        }
    }
    // copy data to d columns
    thrust::device_ptr<int_type> d_col((int_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_int[colname].begin());
	thrust::copy(d_columns_int[colname].begin(), d_columns_int[colname].begin()+recCount, h_columns_int[colname].begin());
}

void CudaSet::addDeviceColumn(float_type* col, string colname, size_t recCount, bool is_decimal) {
    if (std::find(columnNames.begin(), columnNames.end(), colname) == columnNames.end()) {
        columnNames.push_back(colname);
        type[colname] = 1;
        d_columns_float[colname] = thrust::device_vector<float_type>(recCount);
        h_columns_float[colname] = thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >(recCount);
    } else {  // already exists, my need to resize it
        if (d_columns_float[colname].size() < recCount)
            d_columns_float[colname].resize(recCount);
        if (h_columns_float[colname].size() < recCount)
            h_columns_float[colname].resize(recCount);
    }

    decimal[colname] = is_decimal;
    thrust::device_ptr<float_type> d_col((float_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_float[colname].begin());
}

void CudaSet::gpu_perm(queue<string> sf, thrust::device_vector<unsigned int>& permutation) {
	permutation.resize(mRecCount);
	thrust::sequence(permutation.begin(), permutation.begin() + mRecCount, 0, 1);
	unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation.data());
	void* temp;

	CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount*8));
	string sort_type = "ASC";

	while (!sf.empty()) {
		if (type[sf.front()] == 0) {
			update_permutation(d_columns_int[sf.front()], raw_ptr, mRecCount, sort_type, (int_type*)temp, 64);
		} else if (type[sf.front()] == 1) {
			update_permutation(d_columns_float[sf.front()], raw_ptr, mRecCount, sort_type, (float_type*)temp, 64);
		} else {
			thrust::host_vector<unsigned int> permutation_h = permutation;
			char* temp1 = new char[char_size[sf.front()]*mRecCount];
			update_permutation_char_host(h_columns_char[sf.front()], permutation_h.data(), mRecCount, sort_type, temp1, char_size[sf.front()]);
			delete [] temp1;
			permutation = permutation_h;
		}
		sf.pop();
	}
	hipFree(temp);
}

void CudaSet::compress(string file_name, size_t offset, unsigned int check_type, unsigned int check_val, size_t mCount, const bool append) {
    string str(file_name);
    thrust::device_vector<unsigned int> permutation;
	long long int oldCount;
	bool int_check = 0;

    void* d;
    CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));

    total_count = total_count + mCount;
    if (mCount > total_max && op_sort.empty()) {
        total_max = mCount;
    }

	if (!total_segments && append) {
		string s = file_name + "." + columnNames[0] + ".header";
		iFileSystemHandle* f = file_system->open(s.c_str(), "rb");
		if(f) {
			file_system->read((char *)&oldCount, 8, f);
			file_system->read((char *)&total_segments, 4, f);
			file_system->read((char *)&maxRecs, 4, f);
			if (total_max < maxRecs)
				total_max = maxRecs;
			file_system->close(f);
			total_count = oldCount + mCount;
		}
	}
	string s = file_name + ".interval";
	iFileSystemHandle* f = file_system->open(s.c_str(), "rb");
	if (f) {
    	file_system->seek(f, 0, SEEK_END);
        long length = file_system->tell(f);
        file_system->seek(f, 0, SEEK_SET);
		char* buff = new char[length];
		file_system->read(buff, length, f);
		file_system->close(f);
		char* p = strtok(buff, "|");
		string s1(p);
		p = strtok(NULL, "|");
		string s2(p);
		delete [] buff;

		s = file_name + ".key";
		iFileSystemHandle* f1 = file_system->open(s.c_str(), "rb");
		if (f1) {
			file_system->seek(f1, 0, SEEK_END);
			long length = file_system->tell(f1);
			file_system->seek(f1, 0, SEEK_SET);
			buff = new char[length+1];
			buff[length] = 0;
			file_system->read(buff, length, f1);
			file_system->close(f1);
			string s3(buff);
			delete [] buff;
			load_file_name = file_name;
			calc_intervals(s1, s2, s3, total_segments, append);
			int_check = 1;
		}
	}

    if (!op_sort.empty()) { //sort the segment
		gpu_perm(op_sort, permutation);
    }

    // here we need to check for partitions and if partition_count > 0 -> create partitions
    if (mCount < partition_count || partition_count == 0)
        partition_count = 1;
    unsigned int partition_recs = mCount/partition_count;

    if (!op_sort.empty()) {
        if (total_max < partition_recs)
            total_max = partition_recs;
    }

    total_segments++;
    old_segments = total_segments;
    size_t new_offset;
    for (unsigned int i = 0; i < columnNames.size(); i++) {
		std::clock_t start1 = std::clock();
        string colname = columnNames[i];
        str = file_name + "." + colname;
        curr_file = str;
        str += "." + to_string(total_segments-1);
        new_offset = 0;

        if (type[colname] == 0) {
            thrust::device_ptr<int_type> d_col((int_type*)d);
            if (!op_sort.empty()) {
                thrust::gather(permutation.begin(), permutation.end(), d_columns_int[colname].begin(), d_col);

                for (unsigned int p = 0; p < partition_count; p++) {
                    str = file_name + "." + colname;
                    curr_file = str;
                    str += "." + to_string(total_segments-1);
                    if (p < partition_count - 1) {
                        pfor_compress((int_type*)d + new_offset, partition_recs*int_size, str, h_columns_int[colname], 0);
                    } else {
                        pfor_compress((int_type*)d + new_offset, (mCount - partition_recs*p)*int_size, str, h_columns_int[colname], 0);
                    }
                    new_offset = new_offset + partition_recs;
                    total_segments++;
                }
            } else {
				if(!int_check) {
					thrust::copy(h_columns_int[colname].begin() + offset, h_columns_int[colname].begin() + offset + mCount, d_col);
					pfor_compress(d, mCount*int_size, str, h_columns_int[colname], 0);
				} else {
					pfor_compress(thrust::raw_pointer_cast(d_columns_int[colname].data()), mCount*int_size, str, h_columns_int[colname], 0);
				}
            }
        } else if (type[colname] == 1) {
            if (decimal[colname]) {
                thrust::device_ptr<float_type> d_col((float_type*)d);
                if (!op_sort.empty()) {
                    thrust::gather(permutation.begin(), permutation.end(), d_columns_float[colname].begin(), d_col);
                    thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                    thrust::transform(d_col, d_col+mCount, d_col_dec, float_to_long());

                    for (unsigned int p = 0; p < partition_count; p++) {
                        str = file_name + "." + colname;
                        curr_file = str;
                        str += "." + to_string(total_segments-1);
                        if (p < partition_count - 1)
                            pfor_compress((int_type*)d + new_offset, partition_recs*float_size, str, h_columns_float[colname], 1);
                        else
                            pfor_compress((int_type*)d + new_offset, (mCount - partition_recs*p)*float_size, str, h_columns_float[colname], 1);
                        new_offset = new_offset + partition_recs;
                        total_segments++;
                    }
                } else {
                    thrust::copy(h_columns_float[colname].begin() + offset, h_columns_float[colname].begin() + offset + mCount, d_col);
                    thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                    thrust::transform(d_col, d_col+mCount, d_col_dec, float_to_long());
                    pfor_compress(d, mCount*float_size, str, h_columns_float[colname], 1);
                }
            } else { // do not compress -- float
                thrust::device_ptr<float_type> d_col((float_type*)d);
                if (!op_sort.empty()) {
                    thrust::gather(permutation.begin(), permutation.end(), d_columns_float[colname].begin(), d_col);
                    thrust::copy(d_col, d_col+mRecCount, h_columns_float[colname].begin());
                    for (unsigned int p = 0; p < partition_count; p++) {
                        str = file_name + "." + colname;
                        curr_file = str;
                        str += "." + to_string(total_segments-1);
                        unsigned int curr_cnt;
                        if (p < partition_count - 1)
                            curr_cnt = partition_recs;
                        else
                            curr_cnt = mCount - partition_recs*p;

                        iFileSystemHandle* f = file_system->open(str.c_str(), "ab");
                        file_system->write((char *)&curr_cnt, 4, f);
                        file_system->write((char *)(h_columns_float[colname].data() + new_offset), curr_cnt*float_size, f);
                        new_offset = new_offset + partition_recs;
                        unsigned int comp_type = 3;
                        file_system->write((char *)&comp_type, 4, f);
                        file_system->close(f);
                    }
                } else {
                	iFileSystemHandle* f = file_system->open(str.c_str(), "ab");
                	file_system->write((char *)&mCount, 4, f);
                	file_system->write((char *)(h_columns_float[colname].data() + offset), mCount*float_size, f);
                    unsigned int comp_type = 3;
                    file_system->write((char *)&comp_type, 4, f);
                    file_system->close(f);
                }
            }
        } else { //char
			//populate char_hash
			if (append && total_segments == 1) {
				string s = file_name + "." + colname;
				iFileSystemHandle* f = file_system->open(s.c_str(), "rb");
				if (f) {
					char* strings = new char[oldCount*char_size[colname]];
					file_system->read(strings, oldCount*char_size[colname], f);
					file_system->close(f);
					unsigned int ind = std::find(columnNames.begin(), columnNames.end(), colname) - columnNames.begin();
					for (unsigned int z = 0 ; z < oldCount; z++) {
						char_hash[ind][MurmurHash64A(&strings[z*char_size[colname]], char_size[colname], hash_seed)/2] = z;
					}
					delete [] strings;
				}
			}

            if (!op_sort.empty()) {
                unsigned int*  h_permutation = new unsigned int[mRecCount];
                thrust::copy(permutation.begin(), permutation.end(), h_permutation);
                char* t = new char[char_size[colname]*mRecCount];
                apply_permutation_char_host(h_columns_char[colname], h_permutation, mRecCount, t, char_size[colname]);

                delete [] h_permutation;
                thrust::copy(t, t+ char_size[colname]*mRecCount, h_columns_char[colname]);
                delete [] t;
                for (unsigned int p = 0; p < partition_count; p++) {
                    str = file_name + "." + colname;
                    curr_file = str;
                    str += "." + to_string(total_segments-1);

                    if (p < partition_count - 1)
                        compress_char(str, colname, partition_recs, new_offset, total_segments-1);
                    else
                        compress_char(str, colname, mCount - partition_recs*p, new_offset, total_segments-1);
                    new_offset = new_offset + partition_recs;
                    total_segments++;
                }
            } else {
                compress_char(str, colname, mCount, offset, total_segments-1);
            }
        }

        if ((check_type == 1 && fact_file_loaded) || (check_type == 1 && check_val == 0)) {
            if (!op_sort.empty()) {
                writeHeader(file_name, colname, total_segments-1);
            } else {
                writeHeader(file_name, colname, total_segments);
            }
        }
        total_segments = old_segments;
    }

    hipFree(d);
    if (!op_sort.empty()) {
        total_segments = (old_segments-1)+partition_count;
    }
    permutation.resize(0);
    permutation.shrink_to_fit();
}

void CudaSet::calc_intervals(string dt1, string dt2, string index, unsigned int total_segs, bool append) {
	alloced_switch = 1;
	not_compressed = 1;
	thrust::device_vector<unsigned int> permutation;
	thrust::device_vector<int_type> stencil(maxRecs);
	thrust::device_vector<int_type> d_dt2(maxRecs);
	thrust::device_vector<int_type> d_index(maxRecs);
	phase_copy = 0;

	queue<string> sf;
	sf.push(dt1);
	sf.push(index);
	gpu_perm(sf, permutation);

	for (unsigned int i = 0; i < columnNames.size(); i++) {
		if (type[columnNames[i]] == 0) {
			apply_permutation(d_columns_int[columnNames[i]], thrust::raw_pointer_cast(permutation.data()), mRecCount, (int_type*)thrust::raw_pointer_cast(stencil.data()), 0);
		} else {
			unsigned int*  h_permutation = new unsigned int[mRecCount];
			thrust::copy(permutation.begin(), permutation.end(), h_permutation);
			char* t = new char[char_size[columnNames[i]]*mRecCount];
			apply_permutation_char_host(h_columns_char[columnNames[i]], h_permutation, mRecCount, t, char_size[columnNames[i]]);
			delete [] h_permutation;
			thrust::copy(t, t+ char_size[columnNames[i]]*mRecCount, h_columns_char[columnNames[i]]);
			delete [] t;
		}
    }

	if (type[index] == 2) {
		d_columns_int[index] = thrust::device_vector<int_type>(mRecCount);
		h_columns_int[index] = thrust::host_vector<int_type>(mRecCount);
		for (int i = 0; i < mRecCount; i++)
			h_columns_int[index][i] = MurmurHash64A(&h_columns_char[index][i*char_size[index]], char_size[index], hash_seed)/2;
		d_columns_int[index] = h_columns_int[index];
    }

	thrust::counting_iterator<unsigned int> begin(0);
	gpu_interval ff(thrust::raw_pointer_cast(d_columns_int[dt1].data()), thrust::raw_pointer_cast(d_columns_int[dt2].data()), thrust::raw_pointer_cast(d_columns_int[index].data()));
	thrust::for_each(begin, begin + mRecCount - 1, ff);

	auto stack_count = mRecCount;

	if (append) {
		not_compressed = 0;
		size_t mysz = 8;
		if (char_size[index] > int_size)
			mysz = char_size[index];

		if (mysz*maxRecs > alloced_sz) {
			if(alloced_sz) {
				hipFree(alloced_tmp);
			}
			hipMalloc((void **) &alloced_tmp, mysz*maxRecs);
			alloced_sz = mysz*maxRecs;
		}
		thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
		d_columns_int[dt2].resize(0);

		thrust::device_vector<unsigned int> output(stack_count);
		for (int i = 0; i < total_segments; i++) {
			CopyColumnToGpu(dt2, i, 0);
			if (thrust::count(d_col, d_col+mRecCount, 0)) {
				thrust::copy(d_col, d_col+mRecCount, d_dt2.begin());

				if (type[index] == 2) {
					string f1 = load_file_name + "." + index + "." + to_string(i) + ".hash";
					iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
					unsigned int cnt;
					file_system->read(&cnt, 4, f);
					unsigned long long int* buff = new unsigned long long int[cnt];
					file_system->read(buff, cnt*8, f);
					file_system->close(f);
					thrust::copy(buff, buff + cnt, d_index.begin());
					delete [] buff;
				} else {
					CopyColumnToGpu(index, i, 0);
					thrust::copy(d_col, d_col+mRecCount, d_index.begin());
				}

				thrust::lower_bound(d_columns_int[index].begin(), d_columns_int[index].begin()+stack_count, d_index.begin(), d_index.begin() + mRecCount, output.begin());

				gpu_interval_set f(thrust::raw_pointer_cast(d_columns_int[dt1].data()), thrust::raw_pointer_cast(d_dt2.data()),
														 thrust::raw_pointer_cast(d_index.data()), thrust::raw_pointer_cast(d_columns_int[index].data()),
														 thrust::raw_pointer_cast(output.data()));
				thrust::for_each(begin, begin + mRecCount, f);

				string str = load_file_name + "." + dt2 + "." + to_string(i);;
				pfor_compress(thrust::raw_pointer_cast(d_dt2.data()), mRecCount*int_size, str, h_columns_int[dt2], 0);
			}
		}
	}
}

void CudaSet::writeHeader(string file_name, string colname, unsigned int tot_segs) {
    string str = file_name + "." + colname;
    string ff = str;
    str += ".header";
    iFileSystemHandle* f = file_system->open(str.c_str(), "wb");
    file_system->write((char *)&total_count, 8, f);
    file_system->write((char *)&tot_segs, 4, f);
    file_system->write((char *)&total_max, 4, f);
    file_system->write((char *)&cnt_counts[ff], 4, f);
	LOG(logDEBUG) << "HEADER1 " << total_count << " " << tot_segs << " " << total_max;
	file_system->close(f);
}

void CudaSet::reWriteHeader(string file_name, string colname, unsigned int tot_segs, size_t newRecs, size_t maxRecs1) {
    string str = file_name + "." + colname;
    string ff = str;
    str += ".header";
    iFileSystemHandle* f = file_system->open(str.c_str(), "wb");
    file_system->write((char *)&newRecs, 8, f);
    file_system->write((char *)&tot_segs, 4, f);
    file_system->write((char *)&maxRecs1, 4, f);
    LOG(logDEBUG) << "HEADER2 " << newRecs;
    file_system->close(f);
}

void CudaSet::writeSortHeader(string file_name) {
    string str(file_name);
    unsigned int idx;

    if (!op_sort.empty()) {
        str += ".sort";
        iFileSystemHandle* f = file_system->open(str.c_str(), "wb");
        idx = (unsigned int)op_sort.size();
        file_system->write((char *)&idx, 4, f);
        queue<string> os(op_sort);
        while (!os.empty()) {
            if (verbose)
            	LOG(logDEBUG) << "sorted on " << idx;
            idx = os.front().size();
            file_system->write((char *)&idx, 4, f);
            file_system->write(os.front().data(), idx, f);
            os.pop();
        }
        file_system->close(f);
    } else {
        str += ".sort";
        file_system->remove(str.c_str());
    }

	str = file_name;
    if (!op_presort.empty()) {
        str += ".presort";
        iFileSystemHandle* f = file_system->open(str.c_str(), "wb");
        idx = (unsigned int)op_presort.size();
        file_system->write((char *)&idx, 4, f);
        queue<string> os(op_presort);
        while (!os.empty()) {
            idx = os.front().size();
            file_system->write((char *)&idx, 4, f);
            file_system->write(os.front().data(), idx, f);
            os.pop();
        }
        file_system->close(f);
    } else {
        str += ".presort";
        file_system->remove(str.c_str());
    }
}

void CudaSet::Display(unsigned int limit, bool binary, bool term) {
#define MAXCOLS 128
#define MAXFIELDSIZE 1400

    //-- This should/will be converted to an array holding pointers of malloced sized structures--
    char bigbuf[MAXCOLS * MAXFIELDSIZE];
    memset(bigbuf, 0, MAXCOLS * MAXFIELDSIZE);
    char *fields[MAXCOLS];
    const char *dcolumns[MAXCOLS];
    size_t  mCount;         // num records in play
    bool print_all = 0;
    string ss, str;
    int rows = 0;

    if (limit != 0 && limit < mRecCount) {
        mCount = limit;
    } else {
        mCount = mRecCount;
        print_all = 1;
    }

    LOG(logDEBUG) << "mRecCount=" << mRecCount << " mcount = " << mCount << " term " << term <<  " limit=" << limit << " print_all=" << print_all;

    unsigned int cc = 0;
    unordered_map<string, iFileSystemHandle*> file_map;
    unordered_map<string, unsigned int> len_map;

    for (unsigned int i = 0; i < columnNames.size(); i++) {
        fields[cc] = &(bigbuf[cc*MAXFIELDSIZE]);                        // a hack to avoid malloc overheads     - refine later
        dcolumns[cc++] = columnNames[i].c_str();

		if (string_map.find(columnNames[i]) != string_map.end()) {
			auto s = string_map[columnNames[i]];
			auto pos = s.find_first_of(".");
			auto len = data_dict->get_column_length(s.substr(0, pos), s.substr(pos+1));
			iFileSystemHandle* f = file_system->open(string_map[columnNames[i]].c_str(), "rb");
			file_map[string_map[columnNames[i]]] = f;
			len_map[string_map[columnNames[i]]] = len;
		}
    }

    // The goal here is to loop fast and avoid any double handling of outgoing data - pointers are good.
    if (not_compressed && prm_d.size() == 0) {
        for (unsigned int i=0; i < mCount; i++) {                            // for each record
            for (unsigned int j=0; j < columnNames.size(); j++) {                // for each col
                if (type[columnNames[j]] != 1) {
                    if (string_map.find(columnNames[j]) == string_map.end()) {
						if (decimal_zeroes[columnNames[j]]) {
							str = std::to_string(h_columns_int[columnNames[j]][i]);
							LOG(logDEBUG) << "decimals " << columnNames[j] << " " << decimal_zeroes[columnNames[j]] << " " << h_columns_int[columnNames[j]][i];
							while(str.length() <= decimal_zeroes[columnNames[j]])
								str = '0' + str;
							str.insert(str.length()- decimal_zeroes[columnNames[j]], ".");
							sprintf(fields[j], "%s", str.c_str());
						} else {
							if (!ts_cols[columnNames[j]]) {
								sprintf(fields[j], "%lld", (h_columns_int[columnNames[j]])[i] );
							} else {
								time_t ts = (h_columns_int[columnNames[j]][i])/1000;
								auto ti = gmtime(&ts);
								char buffer[30];
								auto rem = (h_columns_int[columnNames[j]][i])%1000;
								strftime(buffer, 30, "%Y-%m-%d %H.%M.%S", ti);
								//fprintf(file_pr, "%s", buffer);
								//fprintf(file_pr, ".%d", rem);
								sprintf(fields[j], "%s.%d", buffer, rem);

								/*time_t tt = h_columns_int[columnNames[j]][i];
								auto ti = localtime(&tt);
								char buffer[10];
								strftime(buffer,80,"%Y-%m-%d", ti);
								sprintf(fields[j], "%s", buffer);
								*/
							}
						}
					} else {
                        file_system->seek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
                        file_system->read(fields[j], len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
                        fields[j][len_map[string_map[columnNames[j]]]] ='\0'; // zero terminate string
                    }
                } else {
                    sprintf(fields[j], "%.2f", (h_columns_float[columnNames[j]])[i]);
                }
            }
            row_cb(mColumnCount, (char **)fields, (char **)dcolumns);
            rows++;
        }
    } else {
        queue<string> op_vx;
        for (unsigned int i = 0; i < columnNames.size(); i++)
            op_vx.push(columnNames[i]);

        if (prm_d.size() || source) {
            allocColumns(this, op_vx);
        }
        unsigned int curr_seg = 0;
        size_t cnt = 0;
        size_t curr_count, sum_printed = 0;
        resize(maxRecs);
        while (sum_printed < mCount || print_all) {
            if (prm_d.size() || source)  {                            // if host arrays are empty
                copyColumns(this, op_vx, curr_seg, cnt);
                size_t olRecs = mRecCount;
                mRecCount = olRecs;
                CopyToHost(0, mRecCount);
                if (sum_printed + mRecCount <= mCount || print_all)
                    curr_count = mRecCount;
                else
                    curr_count = mCount - sum_printed;
            } else {
                curr_count = mCount;
            }

            sum_printed = sum_printed + mRecCount;
            for (unsigned int i=0; i < curr_count; i++) {
                for (unsigned int j=0; j < columnNames.size(); j++) {
                    if (type[columnNames[j]] != 1) {
                        if (string_map.find(columnNames[j]) == string_map.end()) {
                            sprintf(fields[j], "%lld", (h_columns_int[columnNames[j]])[i] );
                        } else {
							file_system->seek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
							file_system->read(fields[j], len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
							fields[j][len_map[string_map[columnNames[j]]]] ='\0'; // zero terminate string
                        }
                    } else {
                        sprintf(fields[j], "%.2f", (h_columns_float[columnNames[j]])[i] );
                    }
                }
                row_cb(mColumnCount, (char **)fields, (char**)dcolumns);
                rows++;
            }
            curr_seg++;
            if (curr_seg == segCount)
                print_all = 0;
        }
    }      // end else
    for (auto it = file_map.begin(); it != file_map.end(); it++)
    	file_system->close(it->second);
}

void CudaSet::Store(const string file_name, const char* sep, const unsigned int limit, const bool binary, const bool append, const bool term) {
    if (mRecCount == 0 && binary == 1 && !term) { // write tails
        for (unsigned int j=0; j < columnNames.size(); j++) {
            writeHeader(file_name, columnNames[j], total_segments);
        }
        return;
    }

    size_t mCount;
    bool print_all = 0;
	string str;

    if (limit != 0 && limit < mRecCount) {
        mCount = limit;
    } else {
        mCount = mRecCount;
        print_all = 1;
    }

    if (binary == 0) {
        unordered_map<string, iFileSystemHandle*> file_map;
        unordered_map<string, unsigned int> len_map;
        string bf;
        unsigned int max_len = 0;
        for (unsigned int j=0; j < columnNames.size(); j++) {
            if (string_map.find(columnNames[j]) != string_map.end()) {
                auto s = string_map[columnNames[j]];
                auto pos = s.find_first_of(".");
                auto len = data_dict->get_column_length(s.substr(0, pos), s.substr(pos+1));
                if (len > max_len)
                    max_len = len;

                iFileSystemHandle* f = file_system->open(string_map[columnNames[j]].c_str(), "rb");
                file_map[string_map[columnNames[j]]] = f;
                len_map[string_map[columnNames[j]]] = len;
            }
        }
        bf.reserve(max_len);
        iFileSystemHandle* file_pr;
        if (!term) {
        	file_pr = file_system->open(file_name.c_str(), "w");
            if (!file_pr)
            	LOG(logERROR) << "Could not open file " << file_name;
        } else {
            //file_pr = stdout; TODO Fix
        }

        if (not_compressed && prm_d.size() == 0) {
            for (unsigned int i=0; i < mCount; i++) {
                for (unsigned int j=0; j < columnNames.size(); j++) {
                    if (type[columnNames[j]] != 1) {
                        if (string_map.find(columnNames[j]) == string_map.end()) {
							if (decimal_zeroes[columnNames[j]]) {
								str = std::to_string(h_columns_int[columnNames[j]][i]);
								LOG(logDEBUG) << "decimals " << columnNames[j] << " " << decimal_zeroes[columnNames[j]] << " " << h_columns_int[columnNames[j]][i];
								while (str.length() <= decimal_zeroes[columnNames[j]])
									str = '0' + str;
								str.insert(str.length()- decimal_zeroes[columnNames[j]], ".");
								file_system->printf(file_pr, "%s", str.c_str());
							} else {
								if (!ts_cols[columnNames[j]]) {
									file_system->printf(file_pr, "%lld", (h_columns_int[columnNames[j]])[i]);
								} else {
									time_t ts = (h_columns_int[columnNames[j]][i])/1000;
									auto ti = gmtime(&ts);
									char buffer[30];
									auto rem = (h_columns_int[columnNames[j]][i])%1000;
									strftime(buffer, 30, "%Y-%m-%d %H.%M.%S", ti);
									file_system->printf(file_pr, "%s", buffer);
									file_system->printf(file_pr, ".%d", rem);
								}
							}
						} else {
                            //fprintf(file_pr, "%.*s", string_hash[columnNames[j]][h_columns_int[columnNames[j]][i]].size(), string_hash[columnNames[j]][h_columns_int[columnNames[j]][i]].c_str());
							file_system->seek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
							file_system->read(&bf[0], len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
							file_system->printf(file_pr, "%.*s", len_map[string_map[columnNames[j]]], bf.c_str());
                        }
                        file_system->puts(sep, file_pr);
                    } else {
                    	file_system->printf(file_pr, "%.2f", (h_columns_float[columnNames[j]])[i]);
                        file_system->puts(sep, file_pr);
                    }
                }
                if (i != mCount -1 )
                	file_system->puts("\n", file_pr);
            }
            if (!term)
                file_system->close(file_pr);
        } else {
		    queue<string> op_vx;
            string ss;
            for (unsigned int j=0; j < columnNames.size(); j++)
                op_vx.push(columnNames[j]);

            if (prm_d.size() || source) {
                allocColumns(this, op_vx);
            }

            unsigned int curr_seg = 0;
            size_t cnt = 0;
            size_t curr_count, sum_printed = 0;
            mRecCount = 0;
            resize(maxRecs);

            while (sum_printed < mCount || print_all) {
                if (prm_d.size() || source)  {
                    copyColumns(this, op_vx, curr_seg, cnt);
                    if (curr_seg == 0) {
                        if (limit != 0 && limit < mRecCount) {
                            mCount = limit;
                            print_all = 0;
                        } else {
                            mCount = mRecCount;
                            print_all = 1;
                        }
                    }

                    // if host arrays are empty
                    size_t olRecs = mRecCount;
                    mRecCount = olRecs;
                    CopyToHost(0, mRecCount);
                    LOG(logDEBUG) << "start " << sum_printed << " " <<  mRecCount << " " <<  mCount;
                    if (sum_printed + mRecCount <= mCount || print_all) {
                        curr_count = mRecCount;
                    } else {
                        curr_count = mCount - sum_printed;
                    }
                } else {
                    curr_count = mCount;
                }

                sum_printed = sum_printed + mRecCount;
                LOG(logDEBUG) << "sum printed " << sum_printed << " " << curr_count << " " << curr_seg;

                for (unsigned int i=0; i < curr_count; i++) {
                    for (unsigned int j=0; j < columnNames.size(); j++) {
                        if (type[columnNames[j]] != 1) {
                            if (string_map.find(columnNames[j]) == string_map.end()) {
								if (decimal_zeroes[columnNames[j]]) {
									str = std::to_string(h_columns_int[columnNames[j]][i]);
									LOG(logDEBUG) << "decimals " << columnNames[j] << " " << decimal_zeroes[columnNames[j]] << " " << h_columns_int[columnNames[j]][i];
									while (str.length() <= decimal_zeroes[columnNames[j]])
										str = '0' + str;
									str.insert(str.length()- decimal_zeroes[columnNames[j]], ".");
									file_system->printf(file_pr, "%s", str.c_str());
								} else {
									if (!ts_cols[columnNames[j]]) {
										file_system->printf(file_pr, "%lld", (h_columns_int[columnNames[j]])[i]);
									} else {
										time_t ts = (h_columns_int[columnNames[j]][i])/1000;
										auto ti = gmtime(&ts);
										char buffer[30];
										auto rem = (h_columns_int[columnNames[j]][i])%1000;
										strftime(buffer, 30, "%Y-%m-%d %H.%M.%S", ti);
										file_system->printf(file_pr, "%s", buffer);
										file_system->printf(file_pr, ".%d", rem);
									}
								}

							} else {
								file_system->seek(file_map[string_map[columnNames[j]]], h_columns_int[columnNames[j]][i] * len_map[string_map[columnNames[j]]], SEEK_SET);
                                file_system->read(&bf[0], len_map[string_map[columnNames[j]]], file_map[string_map[columnNames[j]]]);
                                file_system->printf(file_pr, "%.*s", len_map[string_map[columnNames[j]]], bf.c_str());
                            }
                            file_system->puts(sep, file_pr);
                        } else  {
                        	file_system->printf(file_pr, "%.2f", (h_columns_float[columnNames[j]])[i]);
                            file_system->puts(sep, file_pr);
                        }
                    }
                    if (i != mCount -1 && (curr_seg != segCount || i < curr_count))
                    	file_system->puts("\n", file_pr);
                }
                curr_seg++;
                if (curr_seg == segCount)
                    print_all = 0;
            }
            if (!term) {
            	file_system->close(file_pr);
            }
        }
		for (auto it = file_map.begin(); it != file_map.end(); it++)
			file_system->close(it->second);
    } else {
        //lets update the data dictionary
        for (unsigned int j=0; j < columnNames.size(); j++) {

        	data_dict->set_column_type(file_name, columnNames[j], type[columnNames[j]]);
            if (type[columnNames[j]] != 2) {
				if(decimal[columnNames[j]])
					data_dict->set_column_length(file_name, columnNames[j], decimal_zeroes[columnNames[j]]);
				else if (ts_cols[columnNames[j]])
					data_dict->set_column_length(file_name, columnNames[j], UINT_MAX);
				else
					data_dict->set_column_length(file_name, columnNames[j], 0);
			} else {
				data_dict->set_column_length(file_name, columnNames[j], char_size[columnNames[j]]);
			}
        }
        save_dict = 1;

        if (text_source) {  //writing a binary file using a text file as a source
            compress(file_name, 0, 1, 0, mCount, append);
            for (unsigned int i = 0; i< columnNames.size(); i++)
                if (type[columnNames[i]] == 2)
                    deAllocColumnOnDevice(columnNames[i]);
        } else { //writing a binary file using a binary file as a source
            fact_file_loaded = 1;
            size_t offset = 0;

            if (!not_compressed) { // records are compressed, for example after filter op.
                //decompress to host
                queue<string> op_vx;
                for (unsigned int i = 0; i< columnNames.size(); i++) {
                    op_vx.push(columnNames[i]);
                }

                allocColumns(this, op_vx);
                size_t oldCnt = mRecCount;
                mRecCount = 0;
                resize(oldCnt);
                mRecCount = oldCnt;
                for (unsigned int i = 0; i < segCount; i++) {
                    size_t cnt = 0;
                    copyColumns(this, op_vx, i, cnt);
                    CopyToHost(0, mRecCount);
                    offset = offset + mRecCount;
                    compress(file_name, 0, 0, i - (segCount-1), mRecCount, append);
                }
            } else {
                // now we have decompressed records on the host
                //call setSegments and compress columns in every segment

                segCount = (mRecCount/process_count + 1);
                offset = 0;

                for (unsigned int z = 0; z < segCount; z++) {
                    if (z < segCount-1) {
                        if (mRecCount < process_count) {
                            mCount = mRecCount;
                        } else {
                            mCount = process_count;
                        }
                    } else {
                        mCount = mRecCount - (segCount-1)*process_count;
                    }
                    compress(file_name, offset, 0, z - (segCount-1), mCount, append);
                    offset = offset + mCount;
                }
            }
        }
    }
}


void CudaSet::compress_char(const string file_name, const string colname, const size_t mCount, const size_t offset, const unsigned int segment) {
    unsigned int len = char_size[colname];

    string h_name, i_name, file_no_seg = file_name.substr(0, file_name.find_last_of("."));
    i_name = file_no_seg + "." + to_string(segment) + ".idx";
    h_name = file_no_seg + "." + to_string(segment) + ".hash";
    iFileSystemHandle* b_file;

    iFileSystemHandle* file_h = file_system->open(h_name.c_str(), "wb");
    file_system->write((char *)&mCount, 4, file_h);

	if (segment == 0) {
		b_file = file_system->open(file_no_seg.c_str(), "wb"); //truncate binary
    } else {
    	b_file = file_system->open(file_no_seg.c_str(), "ab"); //append binary
    }

	if (h_columns_int.find(colname) == h_columns_int.end()) {
        h_columns_int[colname] = thrust::host_vector<int_type >(mCount);
	} else {
		if(h_columns_int[colname].size() < mCount)
			h_columns_int[colname].resize(mCount);
	}
    if (d_columns_int.find(colname) == d_columns_int.end()) {
        d_columns_int[colname] = thrust::device_vector<int_type >(mCount);
	} else {
		if (d_columns_int[colname].size() < mCount)
			d_columns_int[colname].resize(mCount);
	}

	size_t  cnt;
	long long int* hash_array = new long long int[mCount];
	map<unsigned long long int, size_t>::iterator iter;
	unsigned int ind = std::find(columnNames.begin(), columnNames.end(), colname) - columnNames.begin();

	for (unsigned int i = 0 ; i < mCount; i++) {
		hash_array[i] = MurmurHash64A(h_columns_char[colname] + (i+offset)*len, len, hash_seed)/2;
		iter = char_hash[ind].find(hash_array[i]);
		if (iter == char_hash[ind].end()) {
			cnt = char_hash[ind].size();
			char_hash[ind][hash_array[i]] = cnt;
			file_system->write((char *)h_columns_char[colname] + (i+offset)*len, len, b_file);
			h_columns_int[colname][i] = cnt;
		} else {
			h_columns_int[colname][i] = iter->second;
		}
	}

	file_system->write((char *)hash_array, 8*mCount, file_h);
	delete [] hash_array;

    thrust::device_vector<int_type> d_col(mCount);
    thrust::copy(h_columns_int[colname].begin(), h_columns_int[colname].begin() + mCount, d_col.begin());
    pfor_compress(thrust::raw_pointer_cast(d_col.data()), mCount*int_size, i_name, h_columns_int[colname], 0);
    file_system->close(file_h);
    file_system->close(b_file);
}

bool first_time = 1;
size_t rec_sz = 0;
size_t process_piece;

bool CudaSet::LoadBigFile(iFileSystemHandle* file_p, thrust::device_vector<char>& d_readbuff, thrust::device_vector<char*>& dest,
							thrust::device_vector<unsigned int>& ind, thrust::device_vector<unsigned int>& dest_len) {
    const char* sep = separator.c_str();
    unsigned int maxx = cols.rbegin()->first;
	map<unsigned int, string>::iterator it;
	bool done = 0;
	std::clock_t start1 = std::clock();

	vector<int> types;
	vector<int> cl;
	types.push_back(0);
	for(int i = 0; i < maxx; i++) {
		auto iter = cols.find(i+1);
		if(iter != cols.end()) {
			types.push_back(type[iter->second]);
			cl.push_back(iter->first-1);
		} else {
			types.push_back(0);
		}
	}

	if (first_time) {
		if(process_count*4 > getFreeMem()) {
			process_piece = getFreeMem()/4;
		} else {
			process_piece = process_count;
		}
		readbuff = new char[process_piece+1];
		d_readbuff.resize(process_piece+1);
		LOG(logDEBUG) << "set a piece to " << process_piece << " " << getFreeMem();
	}

	thrust::device_vector<unsigned int> ind_cnt(1);
	thrust::device_vector<char> sepp(1);
	sepp[0] = *sep;

	long long int total_processed = 0;
	size_t recs_processed = 0;
	bool finished = 0;
	thrust::device_vector<long long int> dev_pos;
	long long int offset;
	unsigned int cnt = 1;
	const unsigned int max_len = 23;

	while (!done) {
		auto rb = file_system->read(readbuff, process_piece, file_p);

		if (readbuff[rb-1] != '\n') {
			rb++;
			readbuff[rb-1] = '\n';
		}

		if (rb < process_piece) {
			done = 1;
			finished = 1;
			file_system->close(file_p);
		}
		if (total_processed >= process_count)
			done = 1;

		thrust::fill(d_readbuff.begin(), d_readbuff.end(), 0);
		thrust::copy(readbuff, readbuff+rb, d_readbuff.begin());

		auto curr_cnt = thrust::count(d_readbuff.begin(), d_readbuff.begin() + rb, '\n') - 1;

		if (recs_processed == 0 && first_time) {
			rec_sz = curr_cnt;
			if(finished)
				rec_sz++;
			total_max = curr_cnt;
		}

		LOG(logDEBUG) << "curr_cnt " << curr_cnt << " Memory: " << getFreeMem();

		if (first_time) {
			for (unsigned int i=0; i < columnNames.size(); i++) {
				auto colname = columnNames[i];
				if (type[colname] == 0) {
					d_columns_int[colname].resize(d_columns_int[colname].size() + rec_sz);
					h_columns_int[colname].resize(h_columns_int[colname].size() + rec_sz);
				} else if (type[colname] == 1) {
					d_columns_float[colname].resize(d_columns_float[colname].size() + rec_sz);
					h_columns_float[colname].resize(h_columns_float[colname].size() + rec_sz);
				} else {
					char* c = new char[cnt*rec_sz*char_size[columnNames[i]]];
					if(recs_processed > 0) {
						memcpy(c, h_columns_char[columnNames[i]], recs_processed*char_size[columnNames[i]]);
						delete [] h_columns_char[columnNames[i]];
					}
					h_columns_char[columnNames[i]] = c;
					if (recs_processed == 0) {
						void* temp;
						CUDA_SAFE_CALL(hipMalloc((void **) &temp, char_size[columnNames[i]]*rec_sz));
						hipMemset(temp, 0, char_size[columnNames[i]]*rec_sz);
						d_columns_char[columnNames[i]] = (char*)temp;
					}
				}

				if (recs_processed == 0) {
					ind[i] = cl[i];
					void* temp;
					if (type[columnNames[i]] != 2) {
						if(!ts_cols[columnNames[i]]) {
							CUDA_SAFE_CALL(hipMalloc((void **) &temp, max_len*rec_sz));
							dest_len[i] = max_len;
						} else {
							CUDA_SAFE_CALL(hipMalloc((void **) &temp, 23*rec_sz));
							dest_len[i] = 23;
						}
					} else {
						CUDA_SAFE_CALL(hipMalloc((void **) &temp, char_size[columnNames[i]]*rec_sz));
						dest_len[i] = char_size[columnNames[i]];
					}
					dest[i] = (char*)temp;
				}
			}
		}

		for (unsigned int i=0; i < columnNames.size(); i++) {
			if(type[columnNames[i]] != 2) {
				hipMemset(dest[i], 0, max_len*rec_sz);
			} else {
				hipMemset(dest[i], 0, char_size[columnNames[i]]*rec_sz);
			}
		}

		if (dev_pos.size() < curr_cnt+1)
			dev_pos.resize(curr_cnt+1);	//avoiding the unnecessary allocs
		dev_pos[0] = -1;
		thrust::copy_if(thrust::make_counting_iterator((unsigned long long int)0), thrust::make_counting_iterator((unsigned long long int)rb-1),
						d_readbuff.begin(), dev_pos.begin()+1, _1 == '\n');

		if (!finished) {
			if (curr_cnt < rec_sz) {
				offset = (dev_pos[curr_cnt] - rb)+1;
				LOG(logDEBUG) << "PATH 1 " << dev_pos[curr_cnt] << " " << offset;
				file_system->seek(file_p, offset, SEEK_CUR);
				total_processed = total_processed + rb + offset;
				mRecCount = curr_cnt;
			} else {
				offset = (dev_pos[rec_sz] - rb)+1;
				LOG(logDEBUG) << "PATH 2 " << dev_pos[rec_sz] << " " << offset;
				file_system->seek(file_p, offset, SEEK_CUR);
				total_processed = total_processed + rb + offset;
				mRecCount = rec_sz;
			}
		} else {
			mRecCount = curr_cnt + 1;
		}

		thrust::counting_iterator<unsigned int> begin(0);
		ind_cnt[0] = mColumnCount;
		parse_functor ff((const char*)thrust::raw_pointer_cast(d_readbuff.data()), (char**)thrust::raw_pointer_cast(dest.data()), thrust::raw_pointer_cast(ind.data()),
						thrust::raw_pointer_cast(ind_cnt.data()), thrust::raw_pointer_cast(sepp.data()), thrust::raw_pointer_cast(dev_pos.data()), thrust::raw_pointer_cast(dest_len.data()));
		thrust::for_each(begin, begin + mRecCount, ff);

		ind_cnt[0] = max_len;
		for (int i =0; i < mColumnCount; i++) {
			if (type[columnNames[i]] == 0) {  //int
				thrust::device_ptr<char> p1((char*)dest[i]);
				if (p1[4] == '-') { //date
					if(!ts_cols[columnNames[i]]) {
						gpu_date date_ff((const char*)dest[i], (long long int*)thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()) + recs_processed);
						thrust::for_each(begin, begin + mRecCount, date_ff);
					} else {
						gpu_tdate date_ff((const char*)dest[i], (long long int*)thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()) + recs_processed);
						thrust::for_each(begin, begin + mRecCount, date_ff);
					}
				} else { //int
					if (decimal[columnNames[i]]) {
						thrust::device_vector<unsigned int> scale(1);
						scale[0] =  decimal_zeroes[columnNames[i]];
						gpu_atold atold((const char*)dest[i], (long long int*)thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()) + recs_processed,
											thrust::raw_pointer_cast(ind_cnt.data()), thrust::raw_pointer_cast(scale.data()));
						thrust::for_each(begin, begin + mRecCount, atold);
					} else {
						gpu_atoll atoll_ff((const char*)dest[i], (long long int*)thrust::raw_pointer_cast(d_columns_int[columnNames[i]].data()) + recs_processed, thrust::raw_pointer_cast(ind_cnt.data()));
						thrust::for_each(begin, begin + mRecCount, atoll_ff);
					}
				}
				thrust::copy(d_columns_int[columnNames[i]].begin() + recs_processed, d_columns_int[columnNames[i]].begin()+recs_processed+mRecCount, h_columns_int[columnNames[i]].begin() + recs_processed);
			} else if (type[columnNames[i]] == 1) {
				gpu_atof atof_ff((const char*)dest[i], (double*)thrust::raw_pointer_cast(d_columns_float[columnNames[i]].data()) + recs_processed, thrust::raw_pointer_cast(ind_cnt.data()));
				thrust::for_each(begin, begin + mRecCount, atof_ff);
				thrust::copy(d_columns_float[columnNames[i]].begin() + recs_processed, d_columns_float[columnNames[i]].begin()+recs_processed+mRecCount, h_columns_float[columnNames[i]].begin() + recs_processed);
			} else {//char is already done
				thrust::device_ptr<char> p1((char*)dest[i]);
				hipMemcpy(h_columns_char[columnNames[i]] + char_size[columnNames[i]]*recs_processed, (void *)dest[i] , char_size[columnNames[i]]*mRecCount, hipMemcpyDeviceToHost);
			}
		}
		recs_processed = recs_processed + mRecCount;
		cnt++;
	}

	if (finished) {
		for (int i =0; i < mColumnCount; i++) {
			if (dest[i]) {
				hipFree(dest[i]);
				dest[i] = nullptr;
			}
		}
		delete [] readbuff;
	}
	LOG(logDEBUG) << "processed recs " << recs_processed << " " << getFreeMem();
	first_time = 0;
	mRecCount = recs_processed;
	return finished;
}

void CudaSet::free() {
    for (unsigned int i = 0; i < columnNames.size(); i++) {
		if (type[columnNames[i]] == 0 && h_columns_int[columnNames[i]].size()) {
			h_columns_int[columnNames[i]].resize(0);
			h_columns_int[columnNames[i]].shrink_to_fit();
		} else {
			h_columns_float[columnNames[i]].resize(0);
			h_columns_float[columnNames[i]].shrink_to_fit();
		}
    }
	if (prm_d.size()) {
		prm_d.resize(0);
		prm_d.shrink_to_fit();
	}
    deAllocOnDevice();
}

bool* CudaSet::logical_and(bool* column1, bool* column2) {
    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());
    thrust::device_free(dev_ptr2);
    return column1;
}

bool* CudaSet::logical_or(bool* column1, bool* column2) {
    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());
    thrust::device_free(dev_ptr2);
    return column1;
}

bool* CudaSet::compare(int_type s, int_type d, int_type op_type) {
    bool res;

    if (op_type == 2) { // >
        if (d > s)
        	res = 1;
        else
        	res = 0;
    } else if (op_type == 1) { // <
        if (d < s)
        	res = 1;
        else
        	res = 0;
    } else if (op_type == 6) { // >=
        if (d >= s)
        	res = 1;
        else
        	res = 0;
    } else if (op_type == 5) { // <=
        if (d <= s)
        	res = 1;
        else
        	res = 0;
    } else if (op_type == 4) {// =
        if (d == s)
        	res = 1;
        else
        	res = 0;
    } else { // !=
        if (d != s)
        	res = 1;
        else
        	res = 0;
    }

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount, res, (bool)0);

    return thrust::raw_pointer_cast(p);
}

bool* CudaSet::compare(float_type s, float_type d, int_type op_type) {
    bool res;

    if (op_type == 2) { // >
        if ((d-s) > EPSILON) res = 1;
        else res = 0;
    } else if (op_type == 1) { // <
        if ((s-d) > EPSILON) res = 1;
        else res = 0;
    } else if (op_type == 6) { // >=
        if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
	} else if (op_type == 5) { // <=
        if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    } else if (op_type == 4) {// =
        if (((d-s) < EPSILON) && ((d-s) > -EPSILON)) res = 1;
        else res = 0;
    } else { // !=
        if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    }
    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount, res, (bool)0);

    return thrust::raw_pointer_cast(p);
}

bool* CudaSet::compare(float_type* column1, float_type d, int_type op_type) {
    thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr(column1);

    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
    else  // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

    return thrust::raw_pointer_cast(res);
}

bool* CudaSet::compare(int_type* column1, int_type d, int_type op_type, unsigned int p1, unsigned int p2) {
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);

	if(p2)
		d = d*(unsigned int)pow(10, p2);

    if (op_type == 2) { // >
		if(!p1)
			thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
		else
			thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
    } else if (op_type == 1) { // <
		if(!p1)
			thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
		else
			thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
    } else if (op_type == 6) { // >=
		if(!p1)
			thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
		else
			thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
	} else if (op_type == 5) { // <=
		if(!p1)
			thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
		else
			thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
	} else if (op_type == 4) { // =
		if(!p1)
			thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
		else
			thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
	} else { // !=
		if(!p1)
			thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());
		else
			thrust::transform(thrust::make_transform_iterator(dev_ptr, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr+mRecCount, power_functor<int_type>(p1)), thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());
	}
    return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(int_type* column1, int_type* column2, int_type op_type, unsigned int p1, unsigned int p2) {
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) { // >
    	if(!p1 && !p2)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
		else if (p1 && p2)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::greater<int_type>());
		else if (p1)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::greater<int_type>());
		else
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::greater<int_type>());
    } else if (op_type == 1) { // <
    	if(!p1 && !p2)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
		else if (p1 && p2)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::less<int_type>());
		else if (p1)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::less<int_type>());
		else
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::less<int_type>());
    } else if (op_type == 6) { // >=
    	if(!p1 && !p2)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
		else if (p1 && p2)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::greater_equal<int_type>());
		else if (p1)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::greater_equal<int_type>());
		else
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::greater_equal<int_type>());
    } else if (op_type == 5) {  // <=
    	if(!p1 && !p2)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
		else if (p1 && p2)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::less_equal<int_type>());
		else if (p1)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::less_equal<int_type>());
		else
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::less_equal<int_type>());
    } else if (op_type == 4) { // =
    	if (!p1 && !p2)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
		else if (p1 && p2)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::equal_to<int_type>());
		else if (p1)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::equal_to<int_type>());
		else
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::equal_to<int_type>());
    } else { // !=
    	if(!p1 && !p2)
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());
		else if (p1 && p2)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::not_equal_to<int_type>());
		else if (p1)
			thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::not_equal_to<int_type>());
		else
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), temp, thrust::not_equal_to<int_type>());
    }

    return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(float_type* column1, float_type* column2, int_type op_type) {
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(float_type* column1, int_type* column2, int_type op_type) {
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr(column2);
    thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    thrust::device_free(dev_ptr2);
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(int_type* column1, float_type* column2, string op_type, bool reverse) {
	if (alloced_mem.empty()) {
		alloc_pool(maxRecs);
	}
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());
    thrust::device_ptr<int_type> dev_ptr(column1);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation
    thrust::device_ptr<float_type> dev_ptr1(column2);

    if (reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    } else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    }
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);
}

int_type* CudaSet::op(int_type* column1, int_type d, string op_type, bool reverse, unsigned int p1, unsigned int p2) {
	if (alloced_mem.empty()) {
		alloc_pool(maxRecs);
	}
	LOG(logDEBUG) << "OP " << d << " " << op_type << " " << p1 << " " << p2;
	thrust::device_ptr<int_type> temp((int_type*)alloced_mem.back());
    thrust::device_ptr<int_type> dev_ptr1(column1);
	unsigned int d1 = d;
	if (p2)
		d = d*(unsigned int)pow(10, p2);

    if (reverse == 0) {
        if (op_type.compare("MUL") == 0) {
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount,  thrust::make_constant_iterator(d1), temp, thrust::multiplies<int_type>());
		} else if (op_type.compare("ADD") == 0) {
			if (!p1)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d*(unsigned int)pow(10, p2)), temp, thrust::plus<int_type>());
			else
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)),  thrust::make_constant_iterator(d), temp, thrust::plus<int_type>());
		} else if (op_type.compare("MINUS") == 0) {
			if (!p1)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d*(unsigned int)pow(10, p2)), temp, thrust::minus<int_type>());
			else
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)),  thrust::make_constant_iterator(d), temp, thrust::minus<int_type>());
		} else {
			if (!p1)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d*(unsigned int)pow(10, p2)), temp, thrust::divides<int_type>());
			else
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)),  thrust::make_constant_iterator(d), temp, thrust::divides<int_type>());
		}
    }  else {
        if (op_type.compare("MUL") == 0) {
			if(!p1)
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
			else
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::multiplies<int_type>());
		} else if (op_type.compare("ADD") == 0) {
			if (!p1)
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
			else
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::plus<int_type>());
		} else if (op_type.compare("MINUS") == 0) {
			if (!p1)
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
			else
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::minus<int_type>());
		} else {
			if (!p1)
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
			else
				thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d) + mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::divides<int_type>());
		}
    }
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);
}

int_type* CudaSet::op(int_type* column1, int_type* column2, string op_type, bool reverse, unsigned int p1, unsigned int p2) {
	if (alloced_mem.empty()) {
		alloc_pool(maxRecs);
	}
	thrust::device_ptr<int_type> temp((int_type*)alloced_mem.back());
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);

	LOG(logDEBUG) << "OP " <<  op_type << " " << p1 << " " << p2 << " " << reverse;

    if (reverse == 0) {
        if (op_type.compare("MUL") == 0) {
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
		} else if (op_type.compare("ADD") == 0) {
			if (!p1 && !p2)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
			else if (p1 && p2)
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::plus<int_type>());
			else if (p1)
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::plus<int_type>());
			else
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::plus<int_type>());

		} else if (op_type.compare("MINUS") == 0) {
			if(!p1 && !p2)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
			else if (p1 && p2)
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::minus<int_type>());
			else if (p1)
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::minus<int_type>());
			else
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::minus<int_type>());

		} else {
			if (!p1 && !p2)
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
			else if (p1 && p2)
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::divides<int_type>());
			else if (p1)
				thrust::transform(thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), thrust::make_transform_iterator(dev_ptr1+mRecCount, power_functor<int_type>(p1)), dev_ptr2, temp, thrust::divides<int_type>());
			else
				thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), temp, thrust::divides<int_type>());
		}
    } else {
        if (op_type.compare("MUL") == 0) {
			thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
		} else if (op_type.compare("ADD") == 0) {
			if (!p1 && !p2)
				thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
			else if (p1 && p2)
				thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::plus<int_type>());
			else if (p1)
				thrust::transform(dev_ptr2, dev_ptr2+mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::plus<int_type>());
			else
				thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), dev_ptr1, temp, thrust::plus<int_type>());

		} else if (op_type.compare("MINUS") == 0) {
			if (!p1 && !p2)
				thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
			else if (p1 && p2)
				thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::minus<int_type>());
			else if (p1)
				thrust::transform(dev_ptr2, dev_ptr2+mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::minus<int_type>());
			else
				thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), dev_ptr1, temp, thrust::minus<int_type>());
		} else {
			if (!p1 && !p2)
				thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
			else if (p1 && p2)
				thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::divides<int_type>());
			else if (p1)
				thrust::transform(dev_ptr2, dev_ptr2+mRecCount, thrust::make_transform_iterator(dev_ptr1, power_functor<int_type>(p1)), temp, thrust::divides<int_type>());
			else
				thrust::transform(thrust::make_transform_iterator(dev_ptr2, power_functor<int_type>(p2)), thrust::make_transform_iterator(dev_ptr2+mRecCount, power_functor<int_type>(p2)), dev_ptr1, temp, thrust::divides<int_type>());
		}
    }
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(float_type* column1, float_type* column2, string op_type, bool reverse) {
	if (alloced_mem.empty()) {
		alloc_pool(maxRecs);
	}
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);

    if (reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
    } else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    }
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(int_type* column1, float_type d, string op_type, bool reverse) {
	if (alloced_mem.empty()) {
		alloc_pool(maxRecs);
	}
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr(column1);
    thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

    if (reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    } else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    }
    thrust::device_free(dev_ptr1);
	alloced_mem.pop_back();
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(float_type* column1, float_type d, string op_type, bool reverse) {
	if (alloced_mem.empty()) {
		alloc_pool(maxRecs);
	}
	thrust::device_ptr<float_type> temp((float_type*)alloced_mem.back());
    thrust::device_ptr<float_type> dev_ptr1(column1);

    if (reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
    } else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    }
	alloced_mem.pop_back();
    return (float_type*)thrust::raw_pointer_cast(temp);
}

char CudaSet::loadIndex(const string index_name, const unsigned int segment) {
    unsigned int bits_encoded, fit_count, vals_count, sz, real_count;
    void* d_str;
    string f1 = index_name + "." + to_string(segment);
    char res;

	//interactive = 0;
    if (interactive) {
        if (index_buffers.find(f1) == index_buffers.end()) {
        	iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
        	file_system->seek(f, 0, SEEK_END);
            long fileSize = file_system->tell(f);
            char* buff;
            hipHostAlloc(&buff, fileSize, hipHostMallocDefault);

            file_system->seek(f, 0, SEEK_SET);
            file_system->read(buff, fileSize, f);
            file_system->close(f);
            index_buffers[f1] = buff;
        }
        sz = ((unsigned int*)index_buffers[f1])[0];

        idx_dictionary_int[index_name].clear();
        for (unsigned int i = 0; i < sz; i++) {
            idx_dictionary_int[index_name][((int_type*)(index_buffers[f1]+4+8*i))[0]] = i;
        }
        vals_count = ((unsigned int*)(index_buffers[f1]+4 +8*sz))[2];
        real_count = ((unsigned int*)(index_buffers[f1]+4 +8*sz))[3];
        mRecCount = real_count;

        if (idx_vals.count(index_name) == 0) {
	        hipMalloc((void **) &d_str, (vals_count+2)*int_size);
			hipMemcpy(d_str, (void *) &((index_buffers[f1]+4 +8*sz)[0]), (vals_count+2)*int_size, hipMemcpyHostToDevice);
			idx_vals[index_name] = (unsigned long long int*)d_str;
		}

    } else {
    	iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
    	file_system->read(&sz, 4, f);
        int_type* d_array = new int_type[sz];
        idx_dictionary_int[index_name].clear();
        file_system->read((void*)d_array, sz*int_size, f);
        for (unsigned int i = 0; i < sz; i++) {
            idx_dictionary_int[index_name][d_array[i]] = i;
        }
        delete [] d_array;

        file_system->read(&fit_count, 4, f);
        file_system->read(&bits_encoded, 4, f);
        file_system->read(&vals_count, 4, f);
        file_system->read(&real_count, 4, f);

        mRecCount = real_count;

        unsigned long long int* int_array = new unsigned long long int[vals_count+2];
        file_system->seek(f, -16 , SEEK_CUR);
        file_system->read((void*)int_array, vals_count*8 + 16, f);
        file_system->read(&res, 1, f);
        file_system->close(f);
        void* d_str;
        hipMalloc((void **) &d_str, (vals_count+2)*int_size);
        hipMemcpy(d_str, (void *) int_array, (vals_count+2)*int_size, hipMemcpyHostToDevice);
        if (idx_vals.count(index_name))
            hipFree(idx_vals[index_name]);
        idx_vals[index_name] = (unsigned long long int*)d_str;
    }
    return res;
}

void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name) { // compressed data for DIM tables
    mColumnCount = (unsigned int)nameRef.size();
    string f1;
    unsigned int cnt;
    char buffer[4000];
    string str;
    not_compressed = 0;
    mRecCount = Recs;
    hostRecCount = Recs;
    totalRecs = Recs;
    load_file_name = file_name;

    f1 = file_name + ".sort";
    iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
    if (f) {
        unsigned int sz, idx;
        file_system->read((char *)&sz, 4, f);
        for (unsigned int j = 0; j < sz; j++) {
        	file_system->read((char *)&idx, 4, f);
        	file_system->read(buffer, idx, f);
            str.assign(buffer, idx);
            sorted_fields.push(str);
            if (verbose)
            	LOG(logDEBUG) << "segment sorted on " << str;
        }
        file_system->close(f);
    }

    f1 = file_name + ".presort";
    f = file_system->open(f1.c_str(), "rb");
    if (f) {
        unsigned int sz, idx;
        file_system->read((char *)&sz, 4, f);
        for (unsigned int j = 0; j < sz; j++) {
        	file_system->read((char *)&idx, 4, f);
        	file_system->read(buffer, idx, f);
            str.assign(buffer, idx);
            presorted_fields.push(str);
            if (verbose)
            	LOG(logDEBUG) << "presorted on " << str;
        }
        file_system->close(f);
    }

    tmp_table = 0;
    filtered = 0;

    for (unsigned int i=0; i < mColumnCount; i++) {
        columnNames.push_back(nameRef.front());
        cols[colsRef.front()] = nameRef.front();

        if (((typeRef.front()).compare("decimal") == 0) || ((typeRef.front()).compare("int") == 0)) {
            f1 = file_name + "." + nameRef.front() + ".0";
            iFileSystemHandle* f = file_system->open(f1.c_str(), "rb");
            if (!f) {
            	LOG(logERROR) << "Couldn't find field " << nameRef.front() << endl;
                exit(0);
            }
            for (unsigned int j = 0; j < 6; j++)
            	file_system->read((char *)&cnt, 4, f);
            file_system->close(f);
            compTypes[nameRef.front()] = cnt;
        }
		if ((typeRef.front()).compare("timestamp") == 0)
			ts_cols[nameRef.front()] = 1;
		else
			ts_cols[nameRef.front()] = 0;

		if ((typeRef.front()).compare("int") == 0 || (typeRef.front()).compare("timestamp") == 0) {
            type[nameRef.front()] = 0;
            decimal[nameRef.front()] = 0;
			decimal_zeroes[nameRef.front()] = 0;
            h_columns_int[nameRef.front()] = thrust::host_vector<int_type, thrust::system::cuda::experimental::pinned_allocator<int_type> >();
            d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
        } else if ((typeRef.front()).compare("float") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 0;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, thrust::system::cuda::experimental::pinned_allocator<float_type> >();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type >();
        } else if ((typeRef.front()).compare("decimal") == 0) {
            type[nameRef.front()] = 0;
            decimal[nameRef.front()] = 1;
			decimal_zeroes[nameRef.front()] = sizeRef.front();
            h_columns_int[nameRef.front()] = thrust::host_vector<int_type, thrust::system::cuda::experimental::pinned_allocator<int_type> >();
            d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
        } else {
            type[nameRef.front()] = 2;
            decimal[nameRef.front()] = 0;
            h_columns_char[nameRef.front()] = nullptr;
            d_columns_char[nameRef.front()] = nullptr;
            char_size[nameRef.front()] = sizeRef.front();
            string_map[nameRef.front()] = file_name + "." + nameRef.front();
        }
        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    }
}



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs) {
    mColumnCount = (unsigned int)nameRef.size();
    tmp_table = 0;
    filtered = 0;
    mRecCount = 0;
    hostRecCount = Recs;
    segCount = 0;

    for (unsigned int i=0; i < mColumnCount; i++) {
        columnNames.push_back(nameRef.front());
        cols[colsRef.front()] = nameRef.front();

		if ((typeRef.front()).compare("timestamp") == 0)
			ts_cols[nameRef.front()] = 1;
		else
			ts_cols[nameRef.front()] = 0;

        if ((typeRef.front()).compare("int") == 0 || (typeRef.front()).compare("timestamp") == 0) {
            type[nameRef.front()] = 0;
            decimal[nameRef.front()] = 0;
			decimal_zeroes[nameRef.front()] = 0;
            h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
        } else if ((typeRef.front()).compare("float") == 0) {
            type[nameRef.front()] = 1;
            decimal[nameRef.front()] = 0;
            h_columns_float[nameRef.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            d_columns_float[nameRef.front()] = thrust::device_vector<float_type>();
        } else if ((typeRef.front()).compare("decimal") == 0) {
            type[nameRef.front()] = 0;
            decimal[nameRef.front()] = 1;
			decimal_zeroes[nameRef.front()] = sizeRef.front();
            h_columns_int[nameRef.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            d_columns_int[nameRef.front()] = thrust::device_vector<int_type>();
        } else {
            type[nameRef.front()] = 2;
            decimal[nameRef.front()] = 0;
            h_columns_char[nameRef.front()] = nullptr;
            d_columns_char[nameRef.front()] = nullptr;
            char_size[nameRef.front()] = sizeRef.front();
        }
        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    }
}

void CudaSet::initialize(const size_t RecordCount, const unsigned int ColumnCount) {
    mRecCount = RecordCount;
    hostRecCount = RecordCount;
    mColumnCount = ColumnCount;
    filtered = 0;
}

void CudaSet::initialize(queue<string> op_sel, const queue<string> op_sel_as) {
    mRecCount = 0;
    mColumnCount = (unsigned int)op_sel.size();
    segCount = 1;
    not_compressed = 1;
    filtered = 0;
    col_aliases = op_sel_as;
    unsigned int i = 0;
    CudaSet *a;
    while (!op_sel.empty()) {
        for (auto it = varNames.begin(); it != varNames.end(); it++) {
            a = it->second;
            if (std::find(a->columnNames.begin(), a->columnNames.end(), op_sel.front()) != a->columnNames.end())
                break;
        }

        type[op_sel.front()] = a->type[op_sel.front()];
        cols[i] = op_sel.front();
        decimal[op_sel.front()] = a->decimal[op_sel.front()];
		decimal_zeroes[op_sel.front()] = a->decimal_zeroes[op_sel.front()];
        columnNames.push_back(op_sel.front());

        if (a->type[op_sel.front()] == 0)  {
            d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
            //h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
            h_columns_int[op_sel.front()] = thrust::host_vector<int_type>();
        } else if (a->type[op_sel.front()] == 1) {
            d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
            //h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
            h_columns_float[op_sel.front()] = thrust::host_vector<float_type>();
        } else {
            h_columns_char[op_sel.front()] = nullptr;
            d_columns_char[op_sel.front()] = nullptr;
            char_size[op_sel.front()] = a->char_size[op_sel.front()];
        }
        i++;
        op_sel.pop();
    }
}

void CudaSet::initialize(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as) {
    mRecCount = 0;
    mColumnCount = 0;
    queue<string> q_cnt(op_sel);
    unsigned int i = 0;
    set<string> field_names;
    while (!q_cnt.empty()) {
        if (std::find(a->columnNames.begin(), a->columnNames.end(), q_cnt.front()) !=  a->columnNames.end() ||
                std::find(b->columnNames.begin(), b->columnNames.end(), q_cnt.front()) !=  b->columnNames.end())  {
            field_names.insert(q_cnt.front());
        }
        q_cnt.pop();
    }
    mColumnCount = (unsigned int)field_names.size();
    maxRecs = b->maxRecs;
    segCount = 1;
    filtered = 0;
    not_compressed = 1;

    col_aliases = op_sel_as;
    i = 0;
    while (!op_sel.empty()) {
        if (std::find(columnNames.begin(), columnNames.end(), op_sel.front()) ==  columnNames.end()) {
            if (std::find(a->columnNames.begin(), a->columnNames.end(), op_sel.front()) !=  a->columnNames.end()) {
                cols[i] = op_sel.front();
                decimal[op_sel.front()] = a->decimal[op_sel.front()];
                columnNames.push_back(op_sel.front());
                type[op_sel.front()] = a->type[op_sel.front()];
				ts_cols[op_sel.front()] = a->ts_cols[op_sel.front()];

                if (a->type[op_sel.front()] == 0)  {
                    d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
                    h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    if (a->string_map.find(op_sel.front()) != a->string_map.end()) {
                        string_map[op_sel.front()] = a->string_map[op_sel.front()];
                    }
					decimal[op_sel.front()] = a->decimal[op_sel.front()];
					decimal_zeroes[op_sel.front()] = a->decimal_zeroes[op_sel.front()];
                } else if (a->type[op_sel.front()] == 1) {
                    d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
                    h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                } else {
                    h_columns_char[op_sel.front()] = nullptr;
                    d_columns_char[op_sel.front()] = nullptr;
                    char_size[op_sel.front()] = a->char_size[op_sel.front()];
                    string_map[op_sel.front()] = a->string_map[op_sel.front()];
                }
                i++;
            } else if (std::find(b->columnNames.begin(), b->columnNames.end(), op_sel.front()) !=  b->columnNames.end()) {
                columnNames.push_back(op_sel.front());
                cols[i] = op_sel.front();
                decimal[op_sel.front()] = b->decimal[op_sel.front()];
                type[op_sel.front()] = b->type[op_sel.front()];
				ts_cols[op_sel.front()] = b->ts_cols[op_sel.front()];

                if (b->type[op_sel.front()] == 0) {
                    d_columns_int[op_sel.front()] = thrust::device_vector<int_type>();
                    h_columns_int[op_sel.front()] = thrust::host_vector<int_type, pinned_allocator<int_type> >();
                    if (b->string_map.find(op_sel.front()) != b->string_map.end()) {
                        string_map[op_sel.front()] = b->string_map[op_sel.front()];
                    }
					decimal[op_sel.front()] = b->decimal[op_sel.front()];
					decimal_zeroes[op_sel.front()] = b->decimal_zeroes[op_sel.front()];
                }  else if (b->type[op_sel.front()] == 1) {
                    d_columns_float[op_sel.front()] = thrust::device_vector<float_type>();
                    h_columns_float[op_sel.front()] = thrust::host_vector<float_type, pinned_allocator<float_type> >();
                }  else {
                    h_columns_char[op_sel.front()] = nullptr;
                    d_columns_char[op_sel.front()] = nullptr;
                    char_size[op_sel.front()] = b->char_size[op_sel.front()];
                    string_map[op_sel.front()] = b->string_map[op_sel.front()];
                }
                i++;
            }
        }
        op_sel.pop();
    }
}


} // namespace alenka
